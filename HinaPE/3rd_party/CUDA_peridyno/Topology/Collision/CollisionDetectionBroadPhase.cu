#include "hip/hip_runtime.h"
#include "CollisionDetectionBroadPhase.h"

#include "Topology/SparseOctree.h"
#include "Topology/LinearBVH.h"

#include "Timer.h"

#include <thrust/sort.h>

namespace dyno
{
	void print(DArray<int> arr)
	{
		CArray<int> h_arr;
		h_arr.resize(arr.size());

		h_arr.assign(arr);

		for (uint i = 0; i < h_arr.size(); i++)
		{
			printf("%d: %d \n", i, h_arr[i]);
		}

		h_arr.clear();
	};

	void print(DArray<PKey> arr)
	{
		CArray<PKey> h_arr;
		h_arr.resize(arr.size());

		h_arr.assign(arr);

		for (uint i = 0; i < h_arr.size(); i++)
		{
			int id = h_arr[i] & UINT_MAX;
			printf("%d: %d \n", i, id);
		}

		h_arr.clear();
	};

	template<typename TDataType>
	CollisionDetectionBroadPhase<TDataType>::CollisionDetectionBroadPhase()
		: ComputeModule()
	{
		this->varGridSizeLimit()->setValue(0.01);
	}

	template<typename TDataType>
	CollisionDetectionBroadPhase<TDataType>::~CollisionDetectionBroadPhase()
	{
		mH.clear();
		mV0.clear();
		mV1.clear();

		mCounter.clear();
		mNewCounter.clear();

		mIds.clear();
		mKeys.clear();
	}

	template<typename Real, typename Coord>
	__global__ void CDBP_SetupCorners(
		DArray<Real> h,
		DArray<Coord> v0,
		DArray<Coord> v1,
		DArray<AABB> bbox)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);

		if (tId >= bbox.size()) return;

		AABB box = bbox[tId];

		v0[tId] = box.v0;
		v1[tId] = box.v1;

		h[tId] = max(box.v1[0] - box.v0[0], max(box.v1[1] - box.v0[1], box.v1[2] - box.v0[2]));
	}


	template<typename Real>
	__global__ void CDBP_ComputeAABBSize(
		DArray<Real> h,
		DArray<AABB> boundingBox)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);

		if (tId >= boundingBox.size()) return;

		AABB box = boundingBox[tId];

		h[tId] = max(box.v1[0] - box.v0[0], max(box.v1[1] - box.v0[1], box.v1[2] - box.v0[2]));
	}

	template<typename TDataType>
	__global__ void CDBP_RequestIntersectionNumber(
		DArray<int> count,
		DArray<AABB> boundingBox,
		SparseOctree<TDataType> octree)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;

		count[tId] = octree.requestIntersectionNumberFromBottom(boundingBox[tId]);
	}

	template<typename TDataType>
	__global__ void CDBP_RequestIntersectionIds(
		DArray<int> ids,
		DArray<int> count,
		DArray<AABB> boundingBox,
		SparseOctree<TDataType> octree)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;

		octree.reqeustIntersectionIdsFromBottom(ids.begin() + count[tId], boundingBox[tId]);
	}



	template<typename TDataType>
	__global__ void CDBP_RequestIntersectionNumber(
		DArray<int> count,
		DArray<AABB> boundingBox,
		SparseOctree<TDataType> octree,
		bool self_collision)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;
		if (!self_collision)
			count[tId] = octree.requestIntersectionNumberFromBottom(boundingBox[tId]);
		else
			count[tId] = octree.requestIntersectionNumberFromLevel(boundingBox[tId], octree.requestLevelNumber(boundingBox[tId]));
	}

	template<typename TDataType>
	__global__ void CDBP_RequestIntersectionIds(
		DArray<int> ids,
		DArray<int> count,
		DArray<AABB> boundingBox,
		SparseOctree<TDataType> octree,
		bool self_collision)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;

		if (!self_collision)
			octree.reqeustIntersectionIdsFromBottom(ids.begin() + count[tId], boundingBox[tId]);
		else
			octree.reqeustIntersectionIdsFromLevel(ids.begin() + count[tId], boundingBox[tId], octree.requestLevelNumber(boundingBox[tId]));
	}

	template<typename TDataType>
	__global__ void CDBP_RequestIntersectionNumberRemove(
		DArray<uint> count,
		DArray<AABB> boundingBox_src,
		DArray<AABB> boundingBox_tar,
		SparseOctree<TDataType> octree,
		int self_collision)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox_src.size()) return;
		if (self_collision)
			count[tId] = octree.requestIntersectionNumberFromLevel(boundingBox_src[tId], boundingBox_tar.begin(), octree.requestLevelNumber(boundingBox_src[tId]));
		else
			count[tId] = octree.requestIntersectionNumberFromBottom(boundingBox_src[tId], boundingBox_tar.begin());
	}

	template<typename TDataType>
	__global__ void CDBP_RequestIntersectionIdsRemove(
		DArray<int> ids,
		DArray<uint> count,
		DArray<AABB> boundingBox_src,
		DArray<AABB> boundingBox_tar,
		SparseOctree<TDataType> octree,
		int self_collision
	)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox_src.size()) return;
		if (self_collision)
			octree.reqeustIntersectionIdsFromLevel(ids.begin() + count[tId], boundingBox_src[tId], boundingBox_tar.begin(), octree.requestLevelNumber(boundingBox_src[tId]));
		else
			octree.reqeustIntersectionIdsFromBottom(ids.begin() + count[tId], boundingBox_src[tId], boundingBox_tar.begin());
	}

	__global__ void CDBP_SetupKeys(
		DArray<PKey> keys,
		DArray<int> ids,
		DArray<uint> count)
	{
		uint tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= count.size()) return;

		int shift = count[tId];
		int total_num = count.size();
		int n = tId == total_num - 1 ? ids.size() - shift : count[tId + 1] - shift;

		for (int i = 0; i < n; i++)
		{
			uint id = ids[shift + i];
			PKey key_hi = tId;
			PKey key_lo = id;
			keys[shift + i] = key_hi << 32 | key_lo;
		}
	}

	template<typename TDataType>
	__global__ void CDBP_CountDuplicativeIds(
		DArray<uint> new_count,
		DArray<PKey> ids,
		DArray<uint> count,
		DArray<AABB> boundingBox,
		SparseOctree<TDataType> octree)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;

		int total_num = boundingBox.size();
		int shift = count[tId];
		int n = tId == total_num - 1 ? ids.size() - count[total_num - 1] : count[tId + 1] - shift;

		int col_num = 0;

		for (int i = 0; i < n; i++)
		{
			uint B_id = ids[shift + i] & UINT_MAX;
			if (i == 0 || B_id != (ids[shift + i - 1] & UINT_MAX))
			{
				col_num++;
			}
		}

		new_count[tId] = col_num;
	}
	template<typename TDataType>
	__global__ void CDBP_CountDuplicativeIds(
		DArray<uint> new_count,
		DArray<PKey> ids,
		DArray<uint> count,
		DArray<AABB> boundingBox,
		SparseOctree<TDataType> octree,
		bool self_collision)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;

		int total_num = boundingBox.size();
		int shift = count[tId];
		int n = tId == total_num - 1 ? ids.size() - count[total_num - 1] : count[tId + 1] - shift;

		int col_num = 0;

		for (int i = 0; i < n; i++)
		{
			uint B_id = ids[shift + i] & UINT_MAX;
			if (i == 0 || B_id != (ids[shift + i - 1] & UINT_MAX))
			{
				if (self_collision)
				{
					if (B_id != tId)
					{
						if (octree.requestLevelNumber(boundingBox[tId]) == octree.requestLevelNumber(boundingBox[B_id]))
						{
							if (B_id > tId)
								col_num++;
						}
						else
							col_num++;
					}
				}
				else
					col_num++;
			}
		}

		new_count[tId] = col_num;
	}

	template<typename TDataType>
	__global__ void CDBP_RemoveDuplicativeIds(
		DArray<int> new_ids,
		DArray<uint> new_count,
		DArray<PKey> ids,
		DArray<uint> count,
		DArray<AABB> boundingBox,
		SparseOctree<TDataType> octree)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;

		int total_num = boundingBox.size();
		int shift = count[tId];
		int n = tId == total_num - 1 ? ids.size() - count[total_num - 1] : count[tId + 1] - shift;

		int col_num = 0;

		int shift_new = new_count[tId];

		for (int i = 0; i < n; i++)
		{
			uint B_id = ids[shift + i] & UINT_MAX;
			if (i == 0 || B_id != (ids[shift + i - 1] & UINT_MAX))
			{
				new_ids[shift_new + col_num] = B_id;
				col_num++;
			}
		}
	}

	template<typename TDataType>
	__global__ void CDBP_RemoveDuplicativeIds(
		DArrayList<int> contactLists,
		DArray<PKey> ids,
		DArray<uint> count,
		DArray<AABB> boundingBox,
		SparseOctree<TDataType> octree,
		bool self_collision)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boundingBox.size()) return;

		int total_num = boundingBox.size();
		int shift = count[tId];
		int n = tId == total_num - 1 ? ids.size() - count[total_num - 1] : count[tId + 1] - shift;

		List<int>& cList_i = contactLists[tId];

		for (int i = 0; i < n; i++)
		{
			uint B_id = ids[shift + i] & UINT_MAX;
			if (i == 0 || B_id != (ids[shift + i - 1] & UINT_MAX))
			{
				if (self_collision)
				{
					if (B_id != tId)
					{

						if (octree.requestLevelNumber(boundingBox[tId]) == octree.requestLevelNumber(boundingBox[B_id]))
						{

							if (B_id > tId)
							{
								cList_i.insert(B_id);
							}
						}
						else
						{
							cList_i.insert(B_id);
						}
					}
				}
				else
				{
					cList_i.insert(B_id);
				}
			}
		}
	}


	__global__ void CDBP_RevertIds(
		DArray<int> elements)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= elements.size()) return;


		int id = elements[tId];
		elements[tId] = -id - 1;
	}

	template<typename TDataType>
	void CollisionDetectionBroadPhase<TDataType>::compute()
	{
		auto type = this->varAccelerationStructure()->getDataPtr()->currentKey();
		switch (type)
		{
		case EStructure::BVH:
			doCollisionWithLinearBVH();
			break;
		case EStructure::Octree:
			doCollisionWithSparseOctree();
			break;
		default:
			break;
		}
	}

	template<typename TDataType>
	void CollisionDetectionBroadPhase<TDataType>::doCollisionWithSparseOctree()
	{
		auto& aabb_src = this->inSource()->getData();
		auto& aabb_tar = this->inTarget()->getData();
		//:dyno::Array<:dyno::AABB, GPU>& aabb_src = this->inSource()->getData();
		//:dyno::Array<:dyno::AABB, GPU>& aabb_tar = this->inTarget()->getData();

		if (this->outContactList()->isEmpty()) {
			this->outContactList()->allocate();
		}

		auto& contacts = this->outContactList()->getData();

		mV0.resize(aabb_tar.size());
		mV1.resize(aabb_tar.size());
		mH.resize(aabb_tar.size());

		cuExecute(aabb_tar.size(),
			CDBP_SetupCorners,
			mH,
			mV0,
			mV1,
			aabb_tar);

		auto min_val = m_reduce_real.minimum(mH.begin(), mH.size());
		auto min_v0 = m_reduce_coord.minimum(mV0.begin(), mV0.size());
		auto max_v1 = m_reduce_coord.maximum(mV1.begin(), mV1.size());

		min_val = max(min_val, this->varGridSizeLimit()->getData());

		SparseOctree<TDataType> octree;
		octree.setSpace(min_v0 - min_val, min_val, max(max_v1[0] - min_v0[0], max(max_v1[1] - min_v0[1], max_v1[2] - min_v0[2])) + 2.0f * min_val);
		octree.construct(aabb_tar);

		mCounter.resize(aabb_src.size());
		cuExecute(aabb_src.size(),
			CDBP_RequestIntersectionNumberRemove,
			mCounter,
			aabb_src,
			aabb_tar,
			octree,
			self_collision
		);

		int total_node_num = thrust::reduce(thrust::device, mCounter.begin(), mCounter.begin() + mCounter.size(), (int)0, thrust::plus<int>());
		thrust::exclusive_scan(thrust::device, mCounter.begin(), mCounter.begin() + mCounter.size(), mCounter.begin());

		mIds.resize(total_node_num);
		/*
		cuExecute(aabb_src.size(),
			CDBP_RequestIntersectionIds,
			ids,
			counter,
			aabb_src,
			octree,
			self_collision);
			*/
		cuExecute(aabb_src.size(),
			CDBP_RequestIntersectionIdsRemove,
			mIds,
			mCounter,
			aabb_src,
			aabb_tar,
			octree,
			self_collision);

		// 		print(counter);
		// 		print(ids);

		mKeys.resize(mIds.size());

		//remove duplicative ids and self id
		cuExecute(mCounter.size(),
			CDBP_SetupKeys,
			mKeys,
			mIds,
			mCounter);

		thrust::sort(thrust::device, mKeys.begin(), mKeys.begin() + mKeys.size());

		mNewCounter.resize(mCounter.size());
		cuExecute(aabb_src.size(),
			CDBP_CountDuplicativeIds,
			mNewCounter,
			mKeys,
			mCounter,
			aabb_src,
			octree,
			self_collision);

		contacts.resize(mNewCounter);

		cuExecute(aabb_src.size(),
			CDBP_RemoveDuplicativeIds,
			contacts,
			mKeys,
			mCounter,
			aabb_src,
			octree,
			self_collision);

		CArrayList<int> hContacts;
		hContacts.assign(contacts);

		octree.release();
	}

	template<typename TDataType, typename AABB>
	__global__ void CDBP_RequestIntersectionNumberBVH(
		DArray<uint> count,
		DArray<AABB> aabbs,
		LinearBVH<TDataType> bvh,
		bool self_collision)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= aabbs.size()) return;

		if (self_collision)
			count[tId] = bvh.requestIntersectionNumber(aabbs[tId], tId);
		else
			count[tId] = bvh.requestIntersectionNumber(aabbs[tId]);
	}


	template<typename TDataType, typename AABB>
	__global__ void CDBP_RequestIntersectionIdsBVH(
		DArrayList<int> idLists,
		DArray<AABB> aabbs,
		LinearBVH<TDataType> bvh,
		bool self_collision)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= aabbs.size()) return;

		if (self_collision)
			bvh.requestIntersectionIds(idLists[tId], aabbs[tId], tId);
		else
			bvh.requestIntersectionIds(idLists[tId], aabbs[tId]);
	}

	template<typename TDataType>
	void CollisionDetectionBroadPhase<TDataType>::doCollisionWithLinearBVH()
	{
		auto& aabb_src = this->inSource()->getData();
		auto& aabb_tar = this->inTarget()->getData();

		if (this->outContactList()->isEmpty()) {
			this->outContactList()->allocate();
		}

		auto& contacts = this->outContactList()->getData();

		LinearBVH<TDataType> bvh;
		bvh.construct(aabb_tar);

		mCounter.resize(aabb_src.size());
		cuExecute(aabb_src.size(),
			CDBP_RequestIntersectionNumberBVH,
			mCounter,
			aabb_src,
			bvh,
			self_collision);

// 		CArray<uint> hCounter;
// 		hCounter.assign(mCounter);
// 		for (int i = 0; i < hCounter.size(); i++)
// 			std::cout << "Num: " << hCounter[i] << std::endl;
// 		hCounter.clear();

		contacts.resize(mCounter);

		cuExecute(aabb_src.size(),
			CDBP_RequestIntersectionIdsBVH,
			contacts,
			aabb_src,
			bvh,
			self_collision);

// 		CArrayList<int> hContacts;
// 		hContacts.assign(contacts);

		bvh.release();
	}

	DEFINE_CLASS(CollisionDetectionBroadPhase);
}