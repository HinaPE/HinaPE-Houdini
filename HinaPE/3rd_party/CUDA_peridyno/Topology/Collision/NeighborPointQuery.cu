#include "hip/hip_runtime.h"
#include "NeighborPointQuery.h"
#include "Topology/GridHash.h"

#include "SceneGraph.h"

namespace dyno
{
	__constant__ int offset_nq[27][3] = { 
		0, 0, 0,
		0, 0, 1,
		0, 1, 0,
		1, 0, 0,
		0, 0, -1,
		0, -1, 0,
		-1, 0, 0,
		0, 1, 1,
		0, 1, -1,
		0, -1, 1,
		0, -1, -1,
		1, 0, 1,
		1, 0, -1,
		-1, 0, 1,
		-1, 0, -1,
		1, 1, 0,
		1, -1, 0,
		-1, 1, 0,
		-1, -1, 0,
		1, 1, 1,
		1, 1, -1,
		1, -1, 1,
		-1, 1, 1,
		1, -1, -1,
		-1, 1, -1,
		-1, -1, 1,
		-1, -1, -1
	};

	IMPLEMENT_TCLASS(NeighborPointQuery, TDataType)

	template<typename TDataType>
	NeighborPointQuery<TDataType>::NeighborPointQuery()
		: ComputeModule()
	{
		this->inOther()->tagOptional(true);

		this->varSizeLimit()->setRange(0, 100);
	}

	template<typename TDataType>
	NeighborPointQuery<TDataType>::~NeighborPointQuery()
	{
	}

	template<typename TDataType>
	void NeighborPointQuery<TDataType>::compute()
	{
		if (this->varSizeLimit()->getData() <= 0) {
			requestDynamicNeighborIds();
		}
		else {
			requestFixedSizeNeighborIds();
		}
	}

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_CalNeighborSize(
		DArray<uint> count,
		DArray<Coord> position_new,
		DArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position_new.size()) return;

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		int counter = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset_nq[c][0], gId3.y + offset_nq[c][1], gId3.z + offset_nq[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					Real d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						counter++;
					}
				}
			}
		}

		count[pId] = counter;
	}
	

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_GetNeighborElements(
		DArrayList<int> nbrIds,
		DArray<Coord> position_new,
		DArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position_new.size()) return;

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		List<int>& list_i = nbrIds[pId];

		int j = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset_nq[c][0], gId3.y + offset_nq[c][1], gId3.z + offset_nq[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					Real d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						list_i.insert(nbId);
						j++;
					}
				}
			}
		}
	}

	template<typename TDataType>
	void NeighborPointQuery<TDataType>::requestDynamicNeighborIds()
	{
		// Prepare inputs
		auto& points	= this->inPosition()->getData();
		auto& other		= this->inOther()->isEmpty() ? this->inPosition()->getData() : this->inOther()->getData();
		auto h			= this->inRadius()->getData();

		// Prepare outputs
		if (this->outNeighborIds()->isEmpty())
			this->outNeighborIds()->allocate();

		auto& nbrIds = this->outNeighborIds()->getData();

		// Construct hash grid
		Reduction<Coord> reduce;
		Coord hiBound = reduce.maximum(points.begin(), points.size());
		Coord loBound = reduce.minimum(points.begin(), points.size());

		// To avoid particles running out of the simulation domain
		auto scn = this->getSceneGraph();
		if (scn != NULL)
		{
			auto loLimit = scn->getLowerBound();
			auto hiLimit = scn->getUpperBound();

			hiBound = hiBound.minimum(hiLimit);
			loBound = loBound.maximum(loLimit);
		}

		GridHash<TDataType> hashGrid;
		hashGrid.setSpace(h, loBound - Coord(h), hiBound + Coord(h));
		hashGrid.clear();
		hashGrid.construct(points);

		DArray<uint> counter(other.size());
		cuExecute(other.size(),
			K_CalNeighborSize,
			counter,
			other,
			points, 
			hashGrid, 
			h);

		nbrIds.resize(counter);

		cuExecute(other.size(),
			K_GetNeighborElements,
			nbrIds, 
			other,
			points, 
			hashGrid,
			h);

		counter.clear();
		hashGrid.release();
	}
	

	template <typename T> __device__ void inline swap_on_device(T& a, T& b) {
		T c(a); a = b; b = c;
	}

	template <typename T>
	__device__ void heapify_up(int* keys, T* vals, int child)
	{
		int parent = (child - 1) / 2;
		while (child > 0)
		{
			if (vals[child] > vals[parent])
			{
				swap_on_device(vals[child], vals[parent]);
				swap_on_device(keys[child], keys[parent]);

				child = parent;
				parent = (child - 1) / 2;
			}
			else
			{
				break;
			}
		}
	}

	template <typename T>
	__device__ void heapify_down(int* keys, T* vals, int node, int size) {
		int j = node;
		while (true) {
			int left = 2 * j + 1;
			int right = 2 * j + 2;
			int largest = j;
			if (left<size && vals[left]>vals[largest]) {
				largest = left;
			}
			if (right<size && vals[right]>vals[largest]) {
				largest = right;
			}
			if (largest == j) return;
			swap_on_device(vals[j], vals[largest]);
			swap_on_device(keys[j], keys[largest]);
			j = largest;
		}
	}

	template <typename T>
	__device__ void heap_sort(int* keys, T* vals, int size) {
		while (size) {
			swap_on_device(vals[0], vals[size - 1]);
			swap_on_device(keys[0], keys[size - 1]);
			heapify_down(keys, vals, 0, --size);
		}
	}

	template<typename Real, typename Coord, typename TDataType>
	__global__ void K_ComputeNeighborFixed(
		DArrayList<int> nbrIds, 
		DArray<Coord> position_new,
		DArray<Coord> position, 
		GridHash<TDataType> hash, 
		Real h,
		int sizeLimit,
		DArray<int> heapIDs,
		DArray<Real> heapDistance)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= position_new.size()) return;

		//TODO: used shared memory for speedup
		int* ids(heapIDs.begin() + pId * sizeLimit);// = new int[nbrLimit];
		Real* distance(heapDistance.begin() + pId * sizeLimit);// = new Real[nbrLimit];

		for (int i = 0; i < sizeLimit; i++) {
			ids[i] = INT_MAX;
			distance[i] = REAL_MAX;
		}

		Coord pos_ijk = position_new[pId];
		int3 gId3 = hash.getIndex3(pos_ijk);

		int counter = 0;
		for (int c = 0; c < 27; c++)
		{
			int cId = hash.getIndex(gId3.x + offset_nq[c][0], gId3.y + offset_nq[c][1], gId3.z + offset_nq[c][2]);
			if (cId >= 0) {
				int totalNum = hash.getCounter(cId);// min(hash.getCounter(cId), hash.npMax);
				for (int i = 0; i < totalNum; i++) {
					int nbId = hash.getParticleId(cId, i);
					float d_ij = (pos_ijk - position[nbId]).norm();
					if (d_ij < h)
					{
						if (counter < sizeLimit)
						{
							ids[counter] = nbId;
							distance[counter] = d_ij;

							heapify_up(ids, distance, counter);
							counter++;
						}
						else
						{
							if (d_ij < distance[0])
							{
								ids[0] = nbId;
								distance[0] = d_ij;

								heapify_down(ids, distance, 0, counter);
							}
						}
						
					}
				}
			}
		}

		List<int>& list_i = nbrIds[pId];

		heap_sort(ids, distance, counter);
		for (int bId = 0; bId < counter; bId++)
		{
			list_i.insert(ids[bId]);
		}
	}

	template<typename TDataType>
	void NeighborPointQuery<TDataType>::requestFixedSizeNeighborIds()
	{
		// Prepare inputs
		auto& points	= this->inPosition()->getData();
		auto& other		= this->inOther()->isEmpty() ? this->inPosition()->getData() : this->inOther()->getData();
		auto h			= this->inRadius()->getData();

		// Prepare outputs
		if (this->outNeighborIds()->isEmpty())
			this->outNeighborIds()->allocate();

		auto& nbrIds = this->outNeighborIds()->getData();

		uint numPt  = this->inPosition()->getDataPtr()->size();
		uint sizeLimit = this->varSizeLimit()->getData();
		
		nbrIds.resize(numPt, sizeLimit);

		// Construct hash grid
		Reduction<Coord> reduce;
		Coord hiBound = reduce.maximum(points.begin(), points.size());
		Coord loBound = reduce.minimum(points.begin(), points.size());

		// To avoid particles running out of the simulation domain
		auto scn = this->getSceneGraph();
		if (scn != NULL)
		{
			auto loLimit = scn->getLowerBound();
			auto hiLimit = scn->getUpperBound();

			hiBound = hiBound.minimum(hiLimit);
			loBound = loBound.maximum(loLimit);
		}


		GridHash<TDataType> hashGrid;
		hashGrid.setSpace(h, loBound - Coord(h), hiBound + Coord(h));
		hashGrid.clear();
		hashGrid.construct(points);

		DArray<int> ids(numPt * sizeLimit);
		DArray<Real> distance(numPt * sizeLimit);
		cuExecute(numPt,
			K_ComputeNeighborFixed,
			nbrIds,
			other,
			points,
			hashGrid,
			h,
			sizeLimit,
			ids,
			distance);

		ids.clear();
		distance.clear();
		//hashGrid.clear();
		hashGrid.release();
	}

	DEFINE_CLASS(NeighborPointQuery);
}