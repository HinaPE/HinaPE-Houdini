#include "hip/hip_runtime.h"
#include "DiscreteElementsToTriangleSet.h"

namespace dyno
{
	typedef typename ::dyno::TOrientedBox3D<Real> Box3D;

	template<typename TDataType>
	DiscreteElementsToTriangleSet<TDataType>::DiscreteElementsToTriangleSet()
		: TopologyMapping()
	{
		mStandardSphere.loadObjFile(getAssetPath() + "standard/standard_icosahedron.obj");
		mStandardCapsule.loadObjFile(getAssetPath() + "standard/standard_capsule.obj");
	}

	template<typename Triangle>
	__global__ void SetupCubeInstances(
		DArray<Vec3f> vertices,
		DArray<Triangle> indices,
		DArray<Box3D> boxes,
		uint pointOffset,
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= boxes.size()) return;
		
		int idx = tId;
		Box3D box = boxes[idx];

		Vec3f hx = box.u * box.extent[0];
		Vec3f hy = box.v * box.extent[1];
		Vec3f hz = box.w * box.extent[2];

		Vec3f hyz = hy + hz;
		Vec3f hxy = hx + hy;
		Vec3f hxz = hx + hz;

		Vec3f c = box.center;

		Vec3f v0 = c - hx - hyz;
		Vec3f v1 = c + hx - hyz;
		Vec3f v2 = c + hxz - hy;
		Vec3f v3 = c - hxy + hz;

		Vec3f v4 = c - hxz + hy;
		Vec3f v5 = c + hxy - hz;
		Vec3f v6 = c + hx + hyz;
		Vec3f v7 = c - hx + hyz;

		vertices[pointOffset + idx * 8] = v0;
		vertices[pointOffset + idx * 8 + 1] = v1;
		vertices[pointOffset + idx * 8 + 2] = v2;
		vertices[pointOffset + idx * 8 + 3] = v3;
		vertices[pointOffset + idx * 8 + 4] = v4;
		vertices[pointOffset + idx * 8 + 5] = v5;
		vertices[pointOffset + idx * 8 + 6] = v6;
		vertices[pointOffset + idx * 8 + 7] = v7;

		uint offset = idx * 8 + pointOffset;

		indices[indexOffset + idx * 12] = Triangle(offset + 0, offset + 1, offset + 2);
		indices[indexOffset + idx * 12 + 1] = Triangle(offset + 0, offset + 2, offset + 3);

		indices[indexOffset + idx * 12 + 2] = Triangle(offset + 0, offset + 4, offset + 5);
		indices[indexOffset + idx * 12 + 3] = Triangle(offset + 0, offset + 5, offset + 1);

		indices[indexOffset + idx * 12 + 4] = Triangle(offset + 4, offset + 7, offset + 6);
		indices[indexOffset + idx * 12 + 5] = Triangle(offset + 4, offset + 6, offset + 5);

		indices[indexOffset + idx * 12 + 6] = Triangle(offset + 1, offset + 5, offset + 6);
		indices[indexOffset + idx * 12 + 7] = Triangle(offset + 1, offset + 6, offset + 2);

		indices[indexOffset + idx * 12 + 8] = Triangle(offset + 2, offset + 6, offset + 7);
		indices[indexOffset + idx * 12 + 9] = Triangle(offset + 2, offset + 7, offset + 3);

		indices[indexOffset + idx * 12 + 10] = Triangle(offset + 0, offset + 3, offset + 7);
		indices[indexOffset + idx * 12 + 11] = Triangle(offset + 0, offset + 7, offset + 4);
	}

	template<typename Triangle>
	__global__ void SetupTetInstances(
		DArray<Vec3f> vertices,
		DArray<Triangle> indices,
		DArray<Tet3D> tets,
		uint pointOffset,
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= tets.size()) return;
	
		int idx = tId;
		Tet3D tet = tets[idx];

		Vec3f v0 = tet.v[0];
		Vec3f v1 = tet.v[1];
		Vec3f v2 = tet.v[2];
		Vec3f v3 = tet.v[3];

		vertices[pointOffset + idx * 4] = v0;
		vertices[pointOffset + idx * 4 + 1] = v1;
		vertices[pointOffset + idx * 4 + 2] = v2;
		vertices[pointOffset + idx * 4 + 3] = v3;

		uint offset = idx * 4 + pointOffset;

		indices[indexOffset + idx * 4] = Triangle(offset + 0, offset + 1, offset + 2);
		indices[indexOffset + idx * 4 + 1] = Triangle(offset + 0, offset + 1, offset + 3);
		indices[indexOffset + idx * 4 + 2] = Triangle(offset + 1, offset + 2, offset + 3);
		indices[indexOffset + idx * 4 + 3] = Triangle(offset + 0, offset + 2, offset + 3);
	}

	__global__ void SetupVerticesForSphereInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> sphereVertices,
		DArray<Sphere3D> sphereInstances,
		uint pointOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInstances.size() * sphereVertices.size()) return;

		uint instanceId = tId / sphereVertices.size();
		uint vertexId = tId % sphereVertices.size();

		Sphere3D sphere = sphereInstances[instanceId];

		Vec3f v = sphereVertices[vertexId];
		vertices[pointOffset + tId] = sphere.center + sphere.radius * sphere.rotation.rotate(v);
	}

	template<typename Triangle>
	__global__ void SetupIndicesForSphereInstances(
		DArray<Triangle> indices,
		DArray<Triangle> sphereIndices,
		DArray<Sphere3D> sphereInstances,
		uint vertexSize,						//vertex size of the instance sphere 
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= sphereInstances.size() * sphereIndices.size()) return;

		uint instanceId = tId / sphereIndices.size();
		uint indexId = tId % sphereIndices.size();

		int vertexOffset = indexOffset + instanceId * vertexSize;
		
		Triangle tIndex = sphereIndices[indexId];
		indices[indexOffset + tId] = Triangle(tIndex[0] + vertexOffset, tIndex[1] + vertexOffset, tIndex[2] + vertexOffset);
	}

	__global__ void SetupRotateForCapsuleInstances(
		DArray<Capsule3D> capsuleInstances,
		DArray<Mat3f> rot)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= capsuleInstances.size()) return;

		Capsule3D capsule = capsuleInstances[tId];
		Vec3f dir = capsule.segment.direction().normalize();
		// Z (0, 0, 1)
		float cos2 = dir[2];
		float cos1 = sqrtf((1 + cos2) / 2.0); 
		float sin1 = sqrtf((1 - cos2) / 2.0);
		Vec3f axis = Vec3f(-dir[1], dir[0], 0).normalize();
		Quat<float> q(axis.x * sin1, axis.y * sin1, axis.z * sin1, cos1);
		if (tId == -1)//DEBUG
		{
			printf("cos (%f %f %f)\n", cos2, cos1, sin1);
			printf("dir (%f %f %f)\n", dir[0], dir[1], dir[2]);
		}
		rot[tId] = q.toMatrix3x3();
	}

	__global__ void SetupVerticesForCapsuleInstances(
		DArray<Vec3f> vertices,
		DArray<Vec3f> capsuleVertices,
		DArray<Capsule3D> capsuleInstances,
		DArray<Mat3f> rot,
		uint pointOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= capsuleInstances.size() * capsuleVertices.size()) return;


		uint instanceId = tId / capsuleVertices.size();
		uint vertexId = tId % capsuleVertices.size();

		Capsule3D capsule = capsuleInstances[instanceId];
		float r = capsule.radius;
		float h = (capsule.segment.length() / 2.0f) - r;
		Vec3f center = (capsule.segment.v0 +  capsule.segment.v1 ) / 2.0f; 

		Vec3f v = capsuleVertices[vertexId];
		Vec3f orignZ = Vec3f(0, 0, 1);
		Vec3f newZ = Vec3f(0, 0, h);
		if (v.z >= 1) // 上半球
		{
			vertices[pointOffset + tId] = rot[instanceId] * ((v - orignZ) * r + newZ) + center;
		}
		else if (v.z <= -1) // 下半球
		{
			vertices[pointOffset + tId] = rot[instanceId] * ((v + orignZ) * r - newZ) + center;
		}
		else // 圆柱
		{
			vertices[pointOffset + tId] = rot[instanceId] * (v * Vec3f(r, r, h)) + center;
		}
		if (tId == -1) //DEBUG
		{
			printf("[%f %f]\n", r, h);
			printf("v: (%f,%f,%f) \n", v.x, v.y, v.z);
			printf("center: (%f,%f,%f) \n", center.x, center.y, center.z);
			printf("ver: (%f,%f,%f) \n", vertices[pointOffset + tId].x, vertices[pointOffset + tId].y, vertices[pointOffset + tId].z);
		}		
	}

	template<typename Triangle>
	__global__ void SetupIndicesForCapsuleInstances(
		DArray<Triangle> indices,
		DArray<Triangle> capsuleIndices,
		DArray<Capsule3D> capsuleInstances,
		uint vertexSize,						//vertex size of the instance sphere 
		uint indexOffset)
	{
		int tId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (tId >= capsuleInstances.size() * capsuleIndices.size()) return;

		uint instanceId = tId / capsuleIndices.size();
		uint indexId = tId % capsuleIndices.size();

		int vertexOffset = indexOffset + instanceId * vertexSize;
		
		Triangle tIndex = capsuleIndices[indexId];
		indices[indexOffset + tId] = Triangle(tIndex[0] + vertexOffset, tIndex[1] + vertexOffset, tIndex[2] + vertexOffset);

		if (tId == -1) //DEBUG
		{
			printf("tIndex [%d %d %d]\n", tIndex[0], tIndex[1], tIndex[2]);
			printf("%d\n", vertexOffset);
		}

	}

	template<typename TDataType>
	bool DiscreteElementsToTriangleSet<TDataType>::apply()
	{
		if (this->outTriangleSet()->isEmpty())
		{
			this->outTriangleSet()->allocate();
		}

		auto inTopo = this->inDiscreteElements()->getDataPtr();

		DArray<Mat3f> capsuleRotates;
		//printf("====================================================== inside box update\n");
		auto& sphereInstances = inTopo->getSpheres();
		auto& capsuleInstances = inTopo->getCaps();
		auto& boxes = inTopo->getBoxes();
		auto& tets = inTopo->getTets();
		auto& tris = inTopo->getTris();
		// TODO : caps

		ElementOffset elementOffset = inTopo->calculateElementOffset();

		int numOfSpheres = sphereInstances.size();
		int numofCaps = capsuleInstances.size();
		int numOfBoxes = boxes.size();
		int numOfTets = tets.size();
		
		auto triSet = this->outTriangleSet()->getDataPtr();

		auto& vertices = triSet->getPoints();
		auto& indices = triSet->getTriangles();

		auto& sphereVertices = mStandardSphere.getPoints();
		auto& sphereIndices = mStandardSphere.getTriangles();

		auto& capsuleVertices = mStandardCapsule.getPoints();
		auto& capsuleIndices = mStandardCapsule.getTriangles();
		
		int numOfVertices = 8 * numOfBoxes + 4 * numOfTets + sphereVertices.size() * numOfSpheres + capsuleVertices.size() * numofCaps;
		int numOfTriangles = 12 * numOfBoxes + 4 * numOfTets + sphereIndices.size() * numOfSpheres + capsuleIndices.size() * numofCaps;

		vertices.resize(numOfVertices);
		indices.resize(numOfTriangles);

		capsuleRotates.resize(numofCaps);

		uint vertexOffset = 0;
		uint indexOffset = 0;

		cuExecute(numOfSpheres * sphereVertices.size(),
			SetupVerticesForSphereInstances,
			vertices,
			sphereVertices,
			sphereInstances,
			vertexOffset);

		cuExecute(numOfSpheres * sphereIndices.size(),
			SetupIndicesForSphereInstances,
			indices,
			sphereIndices,
			sphereInstances,
			sphereVertices.size(),
			indexOffset);

		vertexOffset += numOfSpheres * sphereVertices.size();
		indexOffset += numOfSpheres * sphereIndices.size();
		
		// Capsule
		cuExecute(numofCaps,
			SetupRotateForCapsuleInstances,
			capsuleInstances,
			capsuleRotates);
		
		cuExecute(numofCaps * capsuleVertices.size(),
			SetupVerticesForCapsuleInstances,
			vertices,
			capsuleVertices,
			capsuleInstances,
			capsuleRotates,
			vertexOffset);

		cuExecute(numofCaps * capsuleIndices.size(),
			SetupIndicesForCapsuleInstances,
			indices,
			capsuleIndices,
			capsuleInstances,
			capsuleVertices.size(),
			indexOffset);

		vertexOffset += numofCaps * capsuleVertices.size();
		indexOffset += numofCaps * capsuleIndices.size();

		cuExecute(numOfBoxes,
			SetupCubeInstances,
			vertices,
			indices,
			boxes,
			vertexOffset,
			indexOffset);

		vertexOffset += boxes.size() * 8;
		indexOffset += boxes.size() * 12;

		cuExecute(numOfTets,
			SetupTetInstances,
			vertices,
			indices,
			tets,
			vertexOffset,
			indexOffset);

		this->outTriangleSet()->getDataPtr()->update();

		return true;
	}

	DEFINE_CLASS(DiscreteElementsToTriangleSet);
}