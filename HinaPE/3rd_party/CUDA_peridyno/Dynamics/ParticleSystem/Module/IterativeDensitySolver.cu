#include "hip/hip_runtime.h"
#include "IterativeDensitySolver.h"

#include "SummationDensity.h"

namespace dyno
{
//	IMPLEMENT_TCLASS(DensityPBD, TDataType)

	template<typename TDataType>
	IterativeDensitySolver<TDataType>::IterativeDensitySolver()
		: ParticleApproximation<TDataType>()
	{
		this->varIterationNumber()->setValue(3);
		this->varRestDensity()->setValue(Real(1000));

		mSummation = std::make_shared<SummationDensity<TDataType>>();

		this->inSmoothingLength()->connect(mSummation->inSmoothingLength());
		this->inSamplingDistance()->connect(mSummation->inSamplingDistance());
		this->inPosition()->connect(mSummation->inPosition());
		this->inNeighborIds()->connect(mSummation->inNeighborIds());

		mSummation->outDensity()->connect(this->outDensity());
	}

	template<typename TDataType>
	IterativeDensitySolver<TDataType>::~IterativeDensitySolver()
	{
		mLamda.clear();
		mDeltaPos.clear();
		mPositionOld.clear();
	}


	template <typename Real, typename Coord, typename Kernel>
	__global__ void K_ComputeLambdas(
		DArray<Real> lambdaArr,
		DArray<Real> rhoArr,
		DArray<Coord> posArr,
		DArrayList<int> neighbors,
		Real smoothingLength,
		Kernel gradient,
		Real scale)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		Coord pos_i = posArr[pId];

		Real lamda_i = Real(0);
		Coord grad_ci(0);

		List<int>& list_i = neighbors[pId];
		int nbSize = list_i.size();
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = list_i[ne];
			Real r = (pos_i - posArr[j]).norm();

			if (r > EPSILON)
			{
				Coord g = gradient(r, smoothingLength, scale) * (pos_i - posArr[j]) * (1.0f / r);
				grad_ci += g;
				lamda_i += g.dot(g);
			}
		}

		lamda_i += grad_ci.dot(grad_ci);

		Real rho_i = rhoArr[pId];

		lamda_i = -(rho_i - 1000.0f) / (lamda_i + 0.1f);

		lambdaArr[pId] = lamda_i > 0.0f ? 0.0f : lamda_i;
	}

	template <typename Real, typename Coord, typename Kernel>
	__global__ void K_ComputeDisplacement(
		DArray<Coord> dPos,
		DArray<Real> lambdas,
		DArray<Coord> posArr,
		DArrayList<int> neighbors,
		Real smoothingLength,
		Real dt,
		Kernel gradient,
		Real scale)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		Coord pos_i = posArr[pId];
		Real lamda_i = lambdas[pId];

		Coord dP_i(0);
		List<int>& list_i = neighbors[pId];
		int nbSize = list_i.size();
		for (int ne = 0; ne < nbSize; ne++)
		{
			int j = list_i[ne];
			Real r = (pos_i - posArr[j]).norm();
			if (r > EPSILON)
			{
				Coord dp_ij = 10.0f * (pos_i - posArr[j]) * (lamda_i + lambdas[j]) * gradient(r, smoothingLength, scale) * (1.0 / r);
				dP_i += dp_ij;

				atomicAdd(&dPos[pId][0], dp_ij[0]);
				atomicAdd(&dPos[j][0], -dp_ij[0]);

				if (Coord::dims() >= 2)
				{
					atomicAdd(&dPos[pId][1], dp_ij[1]);
					atomicAdd(&dPos[j][1], -dp_ij[1]);
				}

				if (Coord::dims() >= 3)
				{
					atomicAdd(&dPos[pId][2], dp_ij[2]);
					atomicAdd(&dPos[j][2], -dp_ij[2]);
				}
			}
		}
	}

	template <typename Real, typename Coord>
	__global__ void K_UpdatePosition(
		DArray<Coord> posArr,
		DArray<Coord> velArr,
		DArray<Coord> dPos,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= posArr.size()) return;

		posArr[pId] += dPos[pId];
	}

	template<typename TDataType>
	void IterativeDensitySolver<TDataType>::compute()
	{
		int num = this->inPosition()->size();

		if (mPositionOld.size() != this->inPosition()->size())
			mPositionOld.resize(this->inPosition()->size());

		mPositionOld.assign(this->inPosition()->getData());

		if (this->outDensity()->size() != this->inPosition()->size())
			this->outDensity()->resize(this->inPosition()->size());

		if (mDeltaPos.size() != this->inPosition()->size())
			mDeltaPos.resize(this->inPosition()->size());

		if (mLamda.size() != this->inPosition()->size())
			mLamda.resize(this->inPosition()->size());

		int it = 0;

		int itNum = this->varIterationNumber()->getData();
		while (it < itNum)
		{
			takeOneIteration();

			it++;
		}

		updateVelocity();
	}


	template<typename TDataType>
	void IterativeDensitySolver<TDataType>::takeOneIteration()
	{
		Real dt = this->inTimeStep()->getData();
		int num = this->inPosition()->size();

		mDeltaPos.reset();
		mSummation->varRestDensity()->setValue(this->varRestDensity()->getValue());
		mSummation->varKernelType()->setCurrentKey(this->varKernelType()->currentKey());
		mSummation->update();

		cuFirstOrder(num, this->varKernelType()->getDataPtr()->currentKey(), this->mScalingFactor,
			K_ComputeLambdas,
			mLamda,
			mSummation->outDensity()->getData(),
			this->inPosition()->getData(),
			this->inNeighborIds()->getData(),
			this->inSmoothingLength()->getValue());

		cuFirstOrder(num, this->varKernelType()->getDataPtr()->currentKey(), this->mScalingFactor,
			K_ComputeDisplacement,
			mDeltaPos,
			mLamda,
			this->inPosition()->getData(),
			this->inNeighborIds()->getData(),
			this->inSmoothingLength()->getData(),
			dt);

		cuExecute(num, K_UpdatePosition,
			this->inPosition()->getData(),
			this->inVelocity()->getData(),
			mDeltaPos,
			dt);
	}

	template <typename Real, typename Coord>
	__global__ void DP_UpdateVelocity(
		DArray<Coord> velArr,
		DArray<Coord> curPos,
		DArray<Coord> prePos,
		Real dt)
	{
		int pId = threadIdx.x + (blockIdx.x * blockDim.x);
		if (pId >= velArr.size()) return;

		velArr[pId] += (curPos[pId] - prePos[pId]) / dt;
	}

	template<typename TDataType>
	void IterativeDensitySolver<TDataType>::updateVelocity()
	{
		int num = this->inPosition()->size();

		Real dt = this->inTimeStep()->getData();

		cuExecute(num, DP_UpdateVelocity,
			this->inVelocity()->getData(),
			this->inPosition()->getData(),
			mPositionOld,
			dt);
	}

	DEFINE_CLASS(IterativeDensitySolver);
}