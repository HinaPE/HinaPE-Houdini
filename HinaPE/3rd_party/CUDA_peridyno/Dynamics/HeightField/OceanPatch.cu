#include "hip/hip_runtime.h"
#include "OceanPatch.h"

#include "Topology/HeightField.h"

#include <hip/hip_math_constants.h>

#include <fstream>

namespace dyno
{
    template<typename TDataType>
    OceanPatch<TDataType>::OceanPatch()
        : Node()
    {
        auto heights = std::make_shared<HeightField<TDataType>>();
        this->stateHeightField()->setDataPtr(heights);

        std::ifstream input(getAssetPath() + "windparam.txt", std::ios::in);
        for (int i = 0; i <= 12; i++)
        {
            WindParam param;
            int       dummy;
            input >> dummy;
            input >> param.windSpeed;
            input >> param.A;
            input >> param.choppiness;
            input >> param.global;
            mParams.push_back(param);
        }
        mSpectrumWidth = this->varResolution()->getData() + 1;
        mSpectrumHeight = this->varResolution()->getData() + 4;

        this->varWindDirection()->setRange(0, 360);

        auto callback = std::make_shared<FCallBackFunc>(std::bind(&OceanPatch<TDataType>::resetWindType, this));

        this->varWindType()->attach(callback);
    }

    template<typename TDataType>
    OceanPatch<TDataType>::~OceanPatch()
    {
        mH0.clear();
        mHt.clear();
        mDxt.clear();
        mDzt.clear();
    }

	template<typename Real>
	__device__  Complex<Real> complex_exp(Real arg)
	{
		return Complex<Real>(cosf(arg), sinf(arg));
	}

	// generate wave heightfield at time t based on initial heightfield and dispersion relationship
	template <typename Real, typename Complex>
	__global__ void OP_GenerateSpectrumKernel(
		DArray2D<Complex> h0,
		DArray2D<Complex> ht,
		unsigned int    in_width,
		unsigned int    out_width,
		unsigned int    out_height,
		Real           t,
		Real           patchSize)
	{
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned int in_index = y * in_width + x;
		unsigned int in_mindex = (out_height - y) * in_width + (out_width - x);  // mirrored
		unsigned int out_index = y * out_width + x;

		// calculate wave vector
		Complex k((-(int)out_width / 2.0f + x) * (2.0f * HIP_PI_F / patchSize), (-(int)out_width / 2.0f + y) * (2.0f * HIP_PI_F / patchSize));

		// calculate dispersion w(k)
		Real k_len = k.normSquared();
		Real w = sqrtf(9.81f * k_len);

		if ((x < out_width) && (y < out_height))
		{
			Complex h0_k = h0[in_index];
			Complex h0_mk = h0[in_mindex];

			// output frequency-space complex values
			ht[out_index] = h0_k * complex_exp(w * t) + h0_mk.conjugate() * complex_exp(-w * t);
		}
	}

	template <typename Real, typename Complex>
	__global__ void OP_GenerateDispalcementKernel(
		DArray2D<Complex>      ht,
		DArray2D<Complex>      Dxt,
		DArray2D<Complex>      Dzt,
		unsigned int width,
		unsigned int height,
		Real patchSize)
	{
		unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned int id = y * width + x;

		// calculate wave vector
		Real kx = (-(int)width / 2.0f + x) * (2.0f * HIP_PI_F / patchSize);
		Real ky = (-(int)height / 2.0f + y) * (2.0f * HIP_PI_F / patchSize);
		Real k_squared = kx * kx + ky * ky;
		if (k_squared == 0.0f)
		{
			k_squared = 1.0f;
		}
		kx = kx / sqrtf(k_squared);
		ky = ky / sqrtf(k_squared);

		Complex ht_ij = ht(x, y);
		Complex idoth = Complex(-ht_ij.imagPart(), ht_ij.realPart());

		Dxt(x, y) = kx * idoth;
		Dzt(x, y) = ky * idoth;
	}

    template<typename TDataType>
    void OceanPatch<TDataType>::resetWindType()
    {
        int windType = this->varWindType()->getValue();
        this->varAmplitude()->setValue(mParams[windType].A);
        this->varWindSpeed()->setValue(mParams[windType].windSpeed);
        this->varChoppiness()->setValue(mParams[windType].choppiness);
        this->varGlobalShift()->setValue(mParams[windType].global);
    }

    template<typename TDataType>
    void OceanPatch<TDataType>::resetStates()
    {
        uint res = this->varResolution()->getValue();

        hipfftPlan2d(&fftPlan, res, res, HIPFFT_C2C);

        int spectrumSize = mSpectrumWidth * mSpectrumHeight * sizeof(Complex);
        mH0.resize(mSpectrumWidth, mSpectrumHeight);

        Complex* host_h0 = (Complex*)malloc(spectrumSize);
        generateH0(host_h0);

        cuSafeCall(hipMemcpy(mH0.begin(), host_h0, spectrumSize, hipMemcpyHostToDevice));

        mHt.resize(res, res);
        mDxt.resize(res, res);
        mDzt.resize(res, res);
        this->stateDisplacement()->resize(res, res);

        auto topo = this->stateHeightField()->getDataPtr();
        Real h = this->varPatchSize()->getData() / res;
        topo->setExtents(res, res);
        topo->setGridSpacing(h);
        topo->setOrigin(Vec3f(-0.5 * h * topo->width(), 0, -0.5 * h * topo->height()));

        this->update();
    }

    template<typename TDataType>
    void OceanPatch<TDataType>::updateStates()
    {
        Real timeScaled = this->varTimeScale()->getData() * this->stateElapsedTime()->getData();

        uint res = this->varResolution()->getData();

        cuExecute2D(make_uint2(res, res),
            OP_GenerateSpectrumKernel,
            mH0,
            mHt,
            mSpectrumWidth,
            res,
            res,
            timeScaled,
            this->varPatchSize()->getData());

        cuExecute2D(make_uint2(res, res),
            OP_GenerateDispalcementKernel,
            mHt,
            mDxt,
            mDzt,
            res,
            res,
            this->varPatchSize()->getData());

        hipfftExecC2C(fftPlan, (float2*)mHt.begin(), (float2*)mHt.begin(), HIPFFT_BACKWARD);
        hipfftExecC2C(fftPlan, (float2*)mDxt.begin(), (float2*)mDxt.begin(), HIPFFT_BACKWARD);
        hipfftExecC2C(fftPlan, (float2*)mDzt.begin(), (float2*)mDzt.begin(), HIPFFT_BACKWARD);

        cuExecute2D(make_uint2(res, res),
            O_UpdateDisplacement,
            this->stateDisplacement()->getData(),
            mHt,
            mDxt,
            mDzt,
            res);
    }

    template<typename TDataType>
    void OceanPatch<TDataType>::postUpdateStates()
    {
        auto choppiness = this->varChoppiness()->getValue();

        auto topo = this->stateHeightField()->getDataPtr();

        auto& shifts = topo->getDisplacement();

        uint2 extent;
        extent.x = shifts.nx();
        extent.y = shifts.ny();
        cuExecute2D(extent,
            CW_UpdateHeightDisp,
            shifts,
            this->stateDisplacement()->getData(),
            choppiness);
    }

    template<typename Coord, typename Complex>
    __global__ void O_UpdateDisplacement(
        DArray2D<Coord> displacement,
        DArray2D<Complex> Dh,
        DArray2D<Complex> Dx,
        DArray2D<Complex> Dz,
        int patchSize)
    {
        unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
        if (i < patchSize && j < patchSize)
        {
            Real sign_correction = ((i + j) & 0x01) ? -1.0f : 1.0f;
            Real h_ij = sign_correction * Dh(i, j).realPart();
            Real x_ij = sign_correction * Dx(i, j).realPart();
            Real z_ij = sign_correction * Dz(i, j).realPart();

            displacement(i, j) = Coord(x_ij, h_ij, z_ij);
        }
    }

    template <typename Coord>
    __global__ void CW_UpdateHeightDisp(
        DArray2D<Coord> displacement,
        DArray2D<Coord> dis,
        float choppiness)
    {
        unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
        if (i < displacement.nx() && j < displacement.ny())
        {
            Coord Dij = dis(i, j);

            Coord v;
            v[0] = choppiness * Dij[0];
            v[1] = Dij[1];
            v[2] = choppiness * Dij[2];

            displacement(i, j) = v;
        }
    }

    template<typename TDataType>
    void OceanPatch<TDataType>::generateH0(Complex* h0)
    {
        Real windDir = M_PI * this->varWindDirection()->getValue() / Real(180);
        Real windSpeed = this->varWindSpeed()->getValue();
        Real amplitude = this->varAmplitude()->getValue();

        auto phillips = [=](Real Kx, Real Ky, Real Vdir, Real V, Real A, Real dir_depend) -> Real
        {
            Real k_squared = Kx * Kx + Ky * Ky;

            if (k_squared == 0.0f)
            {
                return 0.0f;
            }

            // largest possible wave from constant wind of velocity v
            Real L = V * V / g;

            Real k_x = Kx / std::sqrt(k_squared);
            Real k_y = Ky / std::sqrt(k_squared);
            Real w_dot_k = k_x * std::cos(Vdir) + k_y * std::sin(Vdir);

            Real phillips = A * std::exp(-1.0f / (k_squared * L * L)) / (k_squared * k_squared) * w_dot_k * w_dot_k;

            // filter out waves moving opposite to wind
            if (w_dot_k < 0.0f)
            {
                phillips *= dir_depend;
            }

            return phillips;
        };

        auto gauss = []() -> Real
        {
            Real u1 = rand() / (Real)RAND_MAX;
            Real u2 = rand() / (Real)RAND_MAX;

            if (u1 < EPSILON)
            {
                u1 = EPSILON;
            }

            return std::sqrt(-2 * std::log(u1)) * std::cos(2 * HIP_PI_F * u2);
        };

        uint res = this->varResolution()->getData();
        for (unsigned int y = 0; y <= res; y++)
        {
            for (unsigned int x = 0; x <= res; x++)
            {
                Real kx = (-(int)res / 2.0f + x) * (2.0f * HIP_PI_F / this->varPatchSize()->getData());
                Real ky = (-(int)res / 2.0f + y) * (2.0f * HIP_PI_F / this->varPatchSize()->getData());

                Real P = std::sqrt(phillips(kx, ky, windDir, windSpeed, amplitude, mDirDepend));

                if (std::abs(kx) < EPSILON && std::abs(ky) == EPSILON)
                {
                    P = 0.0f;
                }

                Real Er = gauss();
                Real Ei = gauss();

                Real h0_re = Er * P * HIP_SQRT_HALF_F;
                Real h0_im = Ei * P * HIP_SQRT_HALF_F;

                int i = y * mSpectrumWidth + x;
                h0[i] = Complex(h0_re, h0_im);
            }
        }
    }

    DEFINE_CLASS(OceanPatch);
}