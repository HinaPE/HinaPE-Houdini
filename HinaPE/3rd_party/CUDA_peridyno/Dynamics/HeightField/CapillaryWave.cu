#include "hip/hip_runtime.h"
﻿#include "CapillaryWave.h"

#include "SceneGraph.h"

#include <Mapping/HeightFieldToTriangleSet.h>

namespace dyno
{
	template<typename TDataType>
	CapillaryWave<TDataType>::CapillaryWave()
		: Node()
	{
		auto heights = std::make_shared<HeightField<TDataType>>();
		this->stateHeightField()->setDataPtr(heights);
	}

	template<typename TDataType>
	CapillaryWave<TDataType>::~CapillaryWave()
	{
		mDeviceGrid.clear();
		mDeviceGridNext.clear();
	}

	template <typename Coord3D, typename Coord4D>
	__global__ void CW_UpdateHeightDisp(
		DArray2D<Coord3D> displacement,
		DArray2D<Coord4D> dis)
	{
		unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
		if (i < displacement.nx() && j < displacement.ny())
		{
			displacement(i, j).y = dis(i, j).x;
		}
	}

	template <typename Coord>
	__device__ float C_GetU(Coord gp)
	{
		Real h = max(gp.x, 0.0f);
		Real uh = gp.y;

		Real h4 = h * h * h * h;
		return sqrtf(2.0f) * h * uh / (sqrtf(h4 + max(h4, EPSILON)));
	}

	template <typename Coord>
	__device__ Real C_GetV(Coord gp)
	{
		Real h = max(gp.x, 0.0f);
		Real vh = gp.z;

		Real h4 = h * h * h * h;
		return sqrtf(2.0f) * h * vh / (sqrtf(h4 + max(h4, EPSILON)));
	}

	template <typename Coord>
	__global__ void CW_MoveSimulatedRegion(
		DArray2D<Coord> grid_next,
		DArray2D<Coord> grid,
		int width,
		int height,
		int dx,
		int dy,
		Real level)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		int j = threadIdx.y + blockIdx.y * blockDim.y;
		if (i < width && j < height)
		{
			int gx = i + 1;
			int gy = j + 1;

			Coord gp = grid(gx, gy);
			Coord gp_init = Coord(level, 0.0f, 0.0f, gp.w);

			int new_i = i - dx;
			int new_j = j - dy;

			if (new_i < 0 || new_i >= width) gp = gp_init;

			new_i = new_i % width;
			if (new_i < 0) new_i = width + new_i;

			if (new_j < 0 || new_j >= height) gp = gp_init;

			new_j = new_j % height;
			if (new_j < 0) new_j = height + new_j;

			grid(new_i + 1, new_j + 1) = gp;
		}
	}

	template<typename TDataType>
	void CapillaryWave<TDataType>::moveDynamicRegion(int nx, int ny)
	{
		auto res = this->varResolution()->getValue();

		auto level = this->varWaterLevel()->getValue();

		int extNx = res + 2;
		int extNy = res + 2;

		cuExecute2D(make_uint2(extNx, extNy),
			CW_MoveSimulatedRegion,
			mDeviceGridNext,
			mDeviceGrid,
			res,
			res,
			nx,
			ny,
			level);

		mOriginX += nx;
		mOriginY += ny;
	}

	template<typename TDataType>
	void CapillaryWave<TDataType>::resetStates()
	{
		int res = this->varResolution()->getValue();
		Real length = this->varLength()->getValue();

		Real level = this->varWaterLevel()->getValue();

		mRealGridSize = length / res;

		int extNx = res + 2;
		int extNy = res + 2;

		mDeviceGrid.resize(extNx, extNy);
		mDeviceGridNext.resize(extNx, extNy);
		this->stateHeight()->resize(res, res);

		//init grid with initial values
		cuExecute2D(make_uint2(extNx, extNy),
			InitDynamicRegion,
			mDeviceGrid,
			extNx,
			extNy,
			level);

		//init grid_next with initial values
		cuExecute2D(make_uint2(extNx, extNy),
			InitDynamicRegion,
			mDeviceGridNext,
			extNx,
			extNy,
			level);

		auto topo = this->stateHeightField()->getDataPtr();
		topo->setExtents(res, res);
		topo->setGridSpacing(mRealGridSize);
		topo->setOrigin(Coord3D(-0.5 * mRealGridSize * topo->width(), 0, -0.5 * mRealGridSize * topo->height()));

		auto& disp = topo->getDisplacement();

		uint2 extent;
		extent.x = disp.nx();
		extent.y = disp.ny();

		cuExecute2D(extent,
			CW_InitHeightDisp,
			this->stateHeight()->getData(),
			disp,
			mDeviceGrid,
			level);
	}

	template<typename TDataType>
	void CapillaryWave<TDataType>::updateStates()
	{
		Real dt = this->stateTimeStep()->getValue();

		Real level = this->varWaterLevel()->getValue();

		uint res = this->varResolution()->getValue();

		int extNx = res + 2;
		int extNy = res + 2;

		int nStep = 1;
		float timestep = dt / nStep;

		auto scn = this->getSceneGraph();
		auto GRAVITY = scn->getGravity().norm();

		for (int iter = 0; iter < nStep; iter++)
		{
			cuExecute2D(make_uint2(extNx, extNy),
				CW_ImposeBC,
				mDeviceGridNext,
				mDeviceGrid,
				extNx,
				extNy);

			cuExecute2D(make_uint2(res, res),
				CW_OneWaveStep,
				mDeviceGrid,
				mDeviceGridNext,
				res,
				res,
				GRAVITY,
				timestep);
		}

		cuExecute2D(make_uint2(res, res),
			CW_InitHeights,
			this->stateHeight()->getData(),
			mDeviceGrid,
			res,
			mRealGridSize);

		cuExecute2D(make_uint2(res, res),
			CW_InitHeightGrad,
			this->stateHeight()->getData(),
			res);

		//Update topology
		auto topo = this->stateHeightField()->getDataPtr();

		auto& disp = topo->getDisplacement();

		uint2 extent;
		extent.x = disp.nx();
		extent.y = disp.ny();

		cuExecute2D(extent,
			CW_UpdateHeightDisp,
			disp,
			this->stateHeight()->getData());
	}

	template <typename Coord4D>
	__global__ void InitDynamicRegion(DArray2D<Coord4D> grid, int gridwidth, int gridheight, float level)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		if (x < gridwidth && y < gridheight)
		{
			Coord4D gp;
			gp.x = level;
			gp.y = 0.0f;
			gp.z = 0.0f;
			gp.w = 0.0f;

			grid(x, y) = gp;
//			if ((x - 256) * (x - 256) + (y - 256) * (y - 256) <= 2500)  grid(x, y).x = level;
		}
	}

	template <typename Coord4D>
	__global__ void CW_ImposeBC(
		DArray2D<Coord4D> grid_next, 
		DArray2D<Coord4D> grid, 
		int width, 
		int height)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;
		if (x < width && y < height)
		{
			if (x == 0)
			{
				Coord4D a = grid(1, y);
				grid_next(x, y) = a;
			}
			else if (x == width - 1)
			{
				Coord4D a = grid(width - 2, y);
				grid_next(x, y) = a;
			}
			else if (y == 0)
			{
				Coord4D a = grid(x, 1);
				grid_next(x, y) = a;
			}
			else if (y == height - 1)
			{
				Coord4D a = grid(x, height - 2);
				grid_next(x, y) = a;
			}
			else
			{
				Coord4D a = grid(x, y);
				grid_next(x, y) = a;
			}
		}
	}

	template <typename Coord>
	__host__ __device__ void CW_FixShore(Coord& l, Coord& c, Coord& r)
	{

		if (r.x < 0.0f || l.x < 0.0f || c.x < 0.0f)
		{
			c.x = c.x + l.x + r.x;
			c.x = max(0.0f, c.x);
			l.x = 0.0f;
			r.x = 0.0f;
		}
		float h = c.x;
		float h4 = h * h * h * h;
		float v = sqrtf(2.0f) * h * c.y / (sqrtf(h4 + max(h4, EPSILON)));
		float u = sqrtf(2.0f) * h * c.z / (sqrtf(h4 + max(h4, EPSILON)));

		c.y = u * h;
		c.z = v * h;
	}

	template <typename Coord>
	__host__ __device__ Coord CW_VerticalPotential(Coord gp, float GRAVITY)
	{
		float h = max(gp.x, 0.0f);
		float uh = gp.y;
		float vh = gp.z;

		float h4 = h * h * h * h;
		float v = sqrtf(2.0f) * h * vh / (sqrtf(h4 + max(h4, EPSILON)));

		Coord G;
		G.x = v * h;
		G.y = uh * v;
		G.z = vh * v + GRAVITY * h * h;
		G.w = 0.0f;
		return G;
	}

	template <typename Coord>
	__device__ Coord CW_HorizontalPotential(Coord gp, float GRAVITY)
	{
		float h = max(gp.x, 0.0f);
		float uh = gp.y;
		float vh = gp.z;

		float h4 = h * h * h * h;
		float u = sqrtf(2.0f) * h * uh / (sqrtf(h4 + max(h4, EPSILON)));

		Coord F;
		F.x = u * h;
		F.y = uh * u + GRAVITY * h * h;
		F.z = vh * u;
		F.w = 0.0f;
		return F;
	}

	template <typename Coord>
	__device__ Coord CW_SlopeForce(Coord c, Coord n, Coord e, Coord s, Coord w, float GRAVITY)
	{
		float h = max(c.x, 0.0f);

		Coord H;
		H.x = 0.0f;
		H.y = -GRAVITY * h * (e.w - w.w);
		H.z = -GRAVITY * h * (s.w - n.w);
		H.w = 0.0f;
		return H;
	}

	template <typename Coord4D>
	__global__ void CW_OneWaveStep(
		DArray2D<Coord4D> grid_next, 
		DArray2D<Coord4D> grid, 
		int width, 
		int height, 
		float GRAVITY, 
		float timestep)
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if (x < width && y < height)
		{
			int gridx = x + 1;
			int gridy = y + 1;

			Coord4D center = grid(gridx, gridy);

			Coord4D north = grid(gridx, gridy - 1);

			Coord4D west = grid(gridx - 1, gridy);

			Coord4D south = grid(gridx, gridy + 1);

			Coord4D east = grid(gridx + 1, gridy);

			CW_FixShore(west, center, east);
			CW_FixShore(north, center, south);

			Coord4D u_south = 0.5f * (south + center) - timestep * (CW_VerticalPotential(south, GRAVITY) - CW_VerticalPotential(center, GRAVITY));
			Coord4D u_north = 0.5f * (north + center) - timestep * (CW_VerticalPotential(center, GRAVITY) - CW_VerticalPotential(north, GRAVITY));
			Coord4D u_west = 0.5f * (west + center) - timestep * (CW_HorizontalPotential(center, GRAVITY) - CW_HorizontalPotential(west, GRAVITY));
			Coord4D u_east = 0.5f * (east + center) - timestep * (CW_HorizontalPotential(east, GRAVITY) - CW_HorizontalPotential(center, GRAVITY));

			Coord4D u_center = center + timestep * CW_SlopeForce(center, north, east, south, west, GRAVITY) - timestep * (CW_HorizontalPotential(u_east, GRAVITY) - CW_HorizontalPotential(u_west, GRAVITY)) - timestep * (CW_VerticalPotential(u_south, GRAVITY) - CW_VerticalPotential(u_north, GRAVITY));
			u_center.x = max(0.0f, u_center.x);

			grid_next(gridx, gridy) = u_center;
		}
	}

	template <typename Coord>
	__global__ void CW_InitHeights(
		DArray2D<Coord> height,
		DArray2D<Coord> grid,
		int patchSize,
		float realSize)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		int j = threadIdx.y + blockIdx.y * blockDim.y;
		if (i < patchSize && j < patchSize)
		{
			int gridx = i + 1;
			int gridy = j + 1;

			Coord gp = grid(gridx, gridy);
			height(i, j) = gp;
		}
	}

	template <typename Coord4D>
	__global__ void CW_InitHeightGrad(
		DArray2D<Coord4D> height,
		int patchSize)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		int j = threadIdx.y + blockIdx.y * blockDim.y;
		if (i < patchSize && j < patchSize)
		{
			int i_minus_one = (i - 1 + patchSize) % patchSize;
			int i_plus_one = (i + 1) % patchSize;
			int j_minus_one = (j - 1 + patchSize) % patchSize;
			int j_plus_one = (j + 1) % patchSize;

			Coord4D Dx = (height(i_plus_one, j) - height(i_minus_one, j)) / 2;
			Coord4D Dz = (height(i, j_plus_one) - height(i, j_minus_one)) / 2;

			height(i, j).z = Dx.y;
			height(i, j).w = Dz.y;
		}
	}

	template <typename Real, typename Coord3D, typename Coord4D>
	__global__ void CW_InitHeightDisp(
		DArray2D<Coord4D> heights,
		DArray2D<Coord3D> displacement,
		DArray2D<Coord4D> grid,
		Real horizon)
	{
		unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
		if (i < displacement.nx() && j < displacement.ny())
		{
			int gridx = i + 1;
			int gridy = j + 1;

			Coord4D gij = grid(gridx, gridy);

			displacement(i, j).x = 0;
			displacement(i, j).y = gij.x + gij.w;
			displacement(i, j).z = 0;

			heights(i, j) = gij;
		}
	}

	DEFINE_CLASS(CapillaryWave);
}