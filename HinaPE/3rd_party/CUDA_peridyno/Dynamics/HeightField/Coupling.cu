#include "hip/hip_runtime.h"
#include "Coupling.h"

#include "Math/Lerp.h"

#include "Primitive/Primitive3D.h"

namespace dyno
{
	template<typename TDataType>
	Coupling<TDataType>::Coupling()
		: Node()
	{
	}

	template<typename TDataType>
	Coupling<TDataType>::~Coupling()
	{
		mForce.clear();
		mTorque.clear();
	}

	template<typename TDataType>
	void Coupling<TDataType>::resetStates()
	{
	}

	template<typename Coord, typename Triangle>
	__global__ void C_ComputeForceAndTorque(
		DArray<Coord> force,
		DArray<Coord> torque,
		DArray<Coord> vertices,
		DArray<Triangle> indices,
		DArray2D<Coord> heights,
		Coord barycenter,
		Coord gravity,
		Coord origin,
		Real spacing,
		Real rho)
	{
		int tId = threadIdx.x + blockIdx.x * blockDim.x;
		if (tId >= indices.size()) return;

		Triangle index_i = indices[tId];

		Coord v0 = vertices[index_i[0]];
		Coord v1 = vertices[index_i[1]];
		Coord v2 = vertices[index_i[2]];

		Triangle3D triangle(v0, v1, v2);

		//Triangle normal
		Coord normal_i = (v2 - v0).cross(v1 - v0);
		normal_i.normalize();

		Coord triangle_center = (v0 + v1 + v2) / Real(3);

		Coord d_i = bilinear(heights, (triangle_center.x - origin.x) / spacing, (triangle_center.z - origin.z) / spacing);

		//Calculate buoyancy
		Real sea_level = d_i.y;
		Real h = triangle_center.y < sea_level ? (sea_level - triangle_center.y) : Real(0);

		Real pressure = rho * gravity.norm() * h;

		Coord force_i = pressure * triangle.area() * normal_i;
		Coord torque_i = -force_i.cross(triangle_center - barycenter);

		force[tId] = force_i;
		torque[tId] = torque_i;
	}

	template<typename TDataType>
	void Coupling<TDataType>::updateStates()
	{
		Real dt = this->stateTimeStep()->getData();

		auto vessels = this->getVessels();
		auto ocean = this->getOcean();

		for (auto mesh : vessels)
		{
			auto& triangles = mesh->stateEnvelope()->getData();

			Real mass = mesh->stateMass()->getData();
			Coord barycenter = mesh->stateBarycenter()->getData();
			Coord velocity = mesh->stateVelocity()->getData();
			Coord angular_velocity = mesh->stateAngularVelocity()->getData();
			Matrix inertia = mesh->stateInertia()->getData();

			Coord gravity = mesh->varGravity()->getData();

			auto& vertices = triangles.getPoints();
			auto& indices = triangles.getTriangles();

			uint num = indices.size();

			if (mForce.size() != num) {
				mForce.resize(num);
				mTorque.resize(num);
			}

			auto heights = ocean->stateHeightField()->getDataPtr();
			auto& displacements = heights->getDisplacement();
			Coord origin = heights->getOrigin();
			Real h = heights->getGridSpacing();

			cuExecute(num,
				C_ComputeForceAndTorque,
				mForce,
				mTorque,
				vertices,
				indices,
				displacements,
				barycenter,
				gravity,
				origin,
				h,
				Real(1000));

			Coord F_total = mReduce.accumulate(mForce.begin(), mForce.size());
			Coord T_total = mReduce.accumulate(mTorque.begin(), mTorque.size());

			velocity += dt * F_total / mass;
			angular_velocity += dt * inertia.inverse() * T_total;

			velocity *= this->varDamping()->getValue();
			angular_velocity *= this->varRotationalDamping()->getValue();

			mesh->stateVelocity()->setValue(velocity);
			mesh->stateAngularVelocity()->setValue(angular_velocity);
		}
	}

	DEFINE_CLASS(Coupling);
}
