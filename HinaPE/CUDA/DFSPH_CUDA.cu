#include "hip/hip_runtime.h"
#include "DFSPH_CUDA.h"

HinaPE_CUDA::DFSPH_AkinciSolverGPU::DFSPH_AkinciSolverGPU(HinaPE_CUDA::real _r, HinaPE_CUDA::Vector _b)
		: NeighborBuilder(_r), MaxBound(_b / 2.), VolumeInited(false)
{

}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::Solve(HinaPE_CUDA::real dt)
{
	build_neighbors();
	compute_density();
	compute_factor();
	divergence_solve(dt);
	non_pressure_force();
	predict_velocity(dt);
	pressure_solve(dt);
	advect(dt);
	enforce_boundary();
}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::build_neighbors() {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::compute_density() {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::compute_factor() {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::divergence_solve(HinaPE_CUDA::real dt) {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::non_pressure_force() {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::predict_velocity(HinaPE_CUDA::real dt) {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::pressure_solve(HinaPE_CUDA::real dt) {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::advect(HinaPE_CUDA::real dt) {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::enforce_boundary() {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_for_each_fluid_particle(const std::function<void(size_t, Vector)> &) {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_for_each_neighbor_fluid(size_t, const std::function<void(size_t, Vector)> &) {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_for_each_neighbor_boundaries(size_t, const std::function<void(size_t, Vector, size_t)> &) {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_resize() {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_compute_akinci_volume() {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_compute_density_change() {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_compute_density_adv(HinaPE_CUDA::real dt) {}
HinaPE_CUDA::real HinaPE_CUDA::DFSPH_AkinciSolverGPU::_compute_density_error(const HinaPE_CUDA::real offset) { return 0; }
HinaPE_CUDA::real HinaPE_CUDA::DFSPH_AkinciSolverGPU::_divergence_solver_iteration(HinaPE_CUDA::real dt) { return 0; }
HinaPE_CUDA::real HinaPE_CUDA::DFSPH_AkinciSolverGPU::_pressure_solve_iteration(HinaPE_CUDA::real dt) { return 0; }
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_divergence_solver_iteration_kernel(HinaPE_CUDA::real dt) {}
void HinaPE_CUDA::DFSPH_AkinciSolverGPU::_pressure_solve_iteration_kernel(HinaPE_CUDA::real dt) {}

__global__ void foo()
{
	printf("I am from CUDA");
}

void HinaPE_CUDA::test1()
{
	foo<<<1, 10>>>();
	hipDeviceSynchronize();
}
