#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/for_each.h>

template <class Func>
__global__ void parallel_for(int n, Func func) {
	for (int i = blockDim.x * blockIdx.x + threadIdx.x;
		 i < n; i += blockDim.x * gridDim.x) {
		func(i);
	}
}

int main() {
	int n = 65536;
	float a = 3.14f;
	thrust::host_vector<float> x_host(n);
	thrust::host_vector<float> y_host(n);

	auto float_rand = [] {
		return std::rand() * (1.f / RAND_MAX);
	};
	thrust::generate(x_host.begin(), x_host.end(), float_rand);
	thrust::generate(y_host.begin(), y_host.end(), float_rand);

	thrust::device_vector<float> x_dev = x_host;
	thrust::device_vector<float> y_dev = x_host;

	thrust::for_each(
			thrust::make_counting_iterator(0),
			thrust::make_counting_iterator(10), [] __device__ (int i) {
				printf("%d\n", i);
			});

	//for (int i = 0; i < n; i++) {
	//printf("x[%d] = %f\n", i, x_host[i]);
	//}

	return 0;
}
