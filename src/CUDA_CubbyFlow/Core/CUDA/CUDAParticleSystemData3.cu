#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include "CUDAParticleSystemData3Func.hpp"

#include <Core/CUDA/CUDAParticleSystemData3.hpp>
#include <Core/CUDA/ThrustUtils.hpp>

#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

using namespace CubbyFlow;

constexpr uint32_t DEFAULT_HASH_GRID_RESOLUTION = 64;

CUDAParticleSystemData3::CUDAParticleSystemData3() : CUDAParticleSystemData3(0)
{
    // Do nothing
}

CUDAParticleSystemData3::CUDAParticleSystemData3(size_t numberOfParticles)
{
    m_positionIdx = AddVectorData();
    m_velocityIdx = AddVectorData();

    Resize(numberOfParticles);
}

CUDAParticleSystemData3::CUDAParticleSystemData3(
    const CUDAParticleSystemData3& other)
{
    Set(other);
}

CUDAParticleSystemData3::CUDAParticleSystemData3(
    CUDAParticleSystemData3&& other) noexcept
    : m_numberOfParticles(std::exchange(other.m_numberOfParticles, 0)),
      m_positionIdx(std::exchange(other.m_positionIdx, 0)),
      m_velocityIdx(std::exchange(other.m_velocityIdx, 0)),
      m_intDataList(std::move(other.m_intDataList)),
      m_floatDataList(std::move(other.m_floatDataList)),
      m_vectorDataList(std::move(other.m_vectorDataList)),
      m_neighborSearcher(std::move(other.m_neighborSearcher)),
      m_neighborStarts(std::move(other.m_neighborStarts)),
      m_neighborEnds(std::move(other.m_neighborEnds)),
      m_neighborLists(std::move(other.m_neighborLists))
{
    // Do nothing
}

CUDAParticleSystemData3& CUDAParticleSystemData3::operator=(
    const CUDAParticleSystemData3& other)
{
    Set(other);
    return *this;
}

CUDAParticleSystemData3& CUDAParticleSystemData3::operator=(
    CUDAParticleSystemData3&& other) noexcept
{
    m_numberOfParticles = std::exchange(other.m_numberOfParticles, 0);
    m_positionIdx = std::exchange(other.m_positionIdx, 0);
    m_velocityIdx = std::exchange(other.m_velocityIdx, 0);

    m_intDataList = std::move(other.m_intDataList);
    m_floatDataList = std::move(other.m_floatDataList);
    m_vectorDataList = std::move(other.m_vectorDataList);

    m_neighborSearcher = std::move(other.m_neighborSearcher);
    m_neighborStarts = std::move(other.m_neighborStarts);
    m_neighborEnds = std::move(other.m_neighborEnds);
    m_neighborLists = std::move(other.m_neighborLists);
    return *this;
}

void CUDAParticleSystemData3::Resize(size_t newNumberOfParticles)
{
    m_numberOfParticles = newNumberOfParticles;

    for (auto& attr : m_intDataList)
    {
        attr.Resize(newNumberOfParticles, 0);
    }

    for (auto& attr : m_floatDataList)
    {
        attr.Resize(newNumberOfParticles, 0.0f);
    }

    for (auto& attr : m_vectorDataList)
    {
        attr.Resize(newNumberOfParticles, make_float4(0, 0, 0, 0));
    }
}

size_t CUDAParticleSystemData3::NumberOfParticles() const
{
    return m_numberOfParticles;
}

size_t CUDAParticleSystemData3::AddIntData(int initialVal)
{
    const size_t attrIdx = m_intDataList.size();
    m_intDataList.emplace_back(NumberOfParticles(), initialVal);
    return attrIdx;
}

size_t CUDAParticleSystemData3::AddFloatData(float initialVal)
{
    const size_t attrIdx = m_floatDataList.size();
    m_floatDataList.emplace_back(NumberOfParticles(), initialVal);
    return attrIdx;
}

size_t CUDAParticleSystemData3::AddVectorData(const Vector4F& initialVal)
{
    const size_t attrIdx = m_vectorDataList.size();
    m_vectorDataList.emplace_back(NumberOfParticles(), ToFloat4(initialVal));
    return attrIdx;
}

size_t CUDAParticleSystemData3::NumberOfIntData() const
{
    return m_intDataList.size();
}

size_t CUDAParticleSystemData3::NumberOfFloatData() const
{
    return m_floatDataList.size();
}

size_t CUDAParticleSystemData3::NumberOfVectorData() const
{
    return m_vectorDataList.size();
}

ConstCUDAArrayView1<float4> CUDAParticleSystemData3::Positions() const
{
    return VectorDataAt(m_positionIdx);
}

CUDAArrayView1<float4> CUDAParticleSystemData3::Positions()
{
    return VectorDataAt(m_positionIdx);
}

ConstCUDAArrayView1<float4> CUDAParticleSystemData3::Velocities() const
{
    return VectorDataAt(m_velocityIdx);
}

CUDAArrayView1<float4> CUDAParticleSystemData3::Velocities()
{
    return VectorDataAt(m_velocityIdx);
}

ConstCUDAArrayView1<int> CUDAParticleSystemData3::IntDataAt(size_t idx) const
{
    return m_intDataList[idx].View();
}

CUDAArrayView1<int> CUDAParticleSystemData3::IntDataAt(size_t idx)
{
    return m_intDataList[idx].View();
}

ConstCUDAArrayView1<float> CUDAParticleSystemData3::FloatDataAt(
    size_t idx) const
{
    return m_floatDataList[idx].View();
}

CUDAArrayView1<float> CUDAParticleSystemData3::FloatDataAt(size_t idx)
{
    return m_floatDataList[idx].View();
}

ConstCUDAArrayView1<float4> CUDAParticleSystemData3::VectorDataAt(
    size_t idx) const
{
    return m_vectorDataList[idx].View();
}

CUDAArrayView1<float4> CUDAParticleSystemData3::VectorDataAt(size_t idx)
{
    return m_vectorDataList[idx].View();
}

void CUDAParticleSystemData3::AddParticle(const Vector4F& newPosition,
                                          const Vector4F& newVelocity)
{
    std::vector<float4> hostPos;
    std::vector<float4> hostVel;
    hostPos.push_back(ToFloat4(newPosition));
    hostVel.push_back(ToFloat4(newVelocity));

    CUDAArray1<float4> devicePos{ hostPos };
    CUDAArray1<float4> deviceVel{ hostVel };

    AddParticles(devicePos, deviceVel);
}

void CUDAParticleSystemData3::AddParticles(
    ConstArrayView1<Vector4F> newPositions,
    ConstArrayView1<Vector4F> newVelocities)
{
    std::vector<float4> hostPos(newPositions.Length());
    std::vector<float4> hostVel(newVelocities.Length());

    for (size_t i = 0; i < newPositions.Length(); ++i)
    {
        hostPos[i] = ToFloat4(newPositions[i]);
    }
    for (size_t i = 0; i < newVelocities.Length(); ++i)
    {
        hostVel[i] = ToFloat4(newVelocities[i]);
    }

    CUDAArray1<float4> devicePos{ hostPos };
    CUDAArray1<float4> deviceVel{ hostVel };

    AddParticles(devicePos, deviceVel);
}

void CUDAParticleSystemData3::AddParticles(
    ConstCUDAArrayView1<float4> newPositions,
    ConstCUDAArrayView1<float4> newVelocities)
{
    assert(newVelocities.Length() == newPositions.Length());

    size_t oldNumberOfParticles = NumberOfParticles();
    Resize(oldNumberOfParticles + newPositions.Length());

    auto pos = Positions();
    CUDACopy(newPositions.data(), newPositions.Length(),
             pos.data() + oldNumberOfParticles);

    if (newVelocities.Length() > 0)
    {
        auto vel = Velocities();
        CUDACopy(newVelocities.data(), newVelocities.Length(),
                 vel.data() + oldNumberOfParticles);
    }
}

ConstCUDAArrayView1<uint32_t> CUDAParticleSystemData3::NeighborStarts() const
{
    return m_neighborStarts.View();
}

ConstCUDAArrayView1<uint32_t> CUDAParticleSystemData3::NeighborEnds() const
{
    return m_neighborEnds.View();
}

const CUDAPointHashGridSearcher3* CUDAParticleSystemData3::NeighborSearcher()
    const
{
    return m_neighborSearcher.get();
}

ConstCUDAArrayView1<uint32_t> CUDAParticleSystemData3::NeighborLists() const
{
    return m_neighborLists.View();
}

void CUDAParticleSystemData3::BuildNeighborSearcher(float maxSearchRadius)
{
    if (m_neighborSearcher == nullptr)
    {
        m_neighborSearcher = std::make_shared<CUDAPointHashGridSearcher3>(
            DEFAULT_HASH_GRID_RESOLUTION, DEFAULT_HASH_GRID_RESOLUTION,
            DEFAULT_HASH_GRID_RESOLUTION, 2.0f * maxSearchRadius);
    }

    m_neighborSearcher->Build(Positions());
}

void CUDAParticleSystemData3::BuildNeighborLists(float maxSearchRadius)
{
    m_neighborStarts.Resize(m_numberOfParticles);
    m_neighborEnds.Resize(m_numberOfParticles);

    auto neighborStarts = m_neighborStarts.View();

    // Count nearby points
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(0) + NumberOfParticles(),
        ForEachNeighborFunc<NoOpFunc, CountNearbyPointsFunc>(
            *m_neighborSearcher, maxSearchRadius, Positions().data(),
            NoOpFunc(), CountNearbyPointsFunc(m_neighborStarts.data())));

    // Make start/end point of neighbor list, and allocate neighbor list.
    thrust::inclusive_scan(thrustCBegin(m_neighborStarts),
                           thrustCEnd(m_neighborStarts),
                           thrustBegin(m_neighborEnds));
    thrust::transform(thrustCBegin(m_neighborEnds), thrustCEnd(m_neighborEnds),
                      thrustCBegin(m_neighborStarts),
                      thrustBegin(m_neighborStarts),
                      thrust::minus<unsigned int>());
    size_t rbeginIdx =
        m_neighborEnds.Length() > 0 ? m_neighborEnds.Length() - 1 : 0;
    uint32_t m = m_neighborEnds[rbeginIdx];
    m_neighborLists.Resize(m, 0);

    // Build neighbor lists
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(0) + NumberOfParticles(),
        ForEachNeighborFunc<BuildNeighborListsFunc, NoOpFunc>(
            *m_neighborSearcher, maxSearchRadius, Positions().data(),
            BuildNeighborListsFunc(m_neighborStarts.data(),
                                   m_neighborEnds.data(),
                                   m_neighborLists.data()),
            NoOpFunc()));
}

void CUDAParticleSystemData3::Set(const CUDAParticleSystemData3& other)
{
    m_numberOfParticles = other.m_numberOfParticles;
    m_positionIdx = other.m_positionIdx;
    m_velocityIdx = other.m_velocityIdx;

    m_intDataList = other.m_intDataList;
    m_floatDataList = other.m_floatDataList;
    m_vectorDataList = other.m_vectorDataList;

    if (other.m_neighborSearcher != nullptr)
    {
        m_neighborSearcher = std::make_shared<CUDAPointHashGridSearcher3>(
            *other.m_neighborSearcher);
    }

    m_neighborStarts = other.m_neighborStarts;
    m_neighborEnds = other.m_neighborEnds;
    m_neighborLists = other.m_neighborLists;
}
