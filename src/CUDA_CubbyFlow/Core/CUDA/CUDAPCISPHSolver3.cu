#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Core/CUDA/CUDAPCISPHSolver3.hpp>
#include <Core/CUDA/CUDASPHKernels3.hpp>

#include <thrust/extrema.h>

using namespace CubbyFlow;

namespace
{
class InitializeBuffersAndComputeForces
{
 public:
    inline InitializeBuffersAndComputeForces(
        float m, float h, float4 gravity, float viscosity,
        const uint32_t* neighborStarts, const uint32_t* neighborEnds,
        const uint32_t* neighborLists, const float4* positions,
        const float4* velocities, float4* smoothedVelocities, float4* forces,
        const float* densities, float* pressures, float4* pressureForces,
        float* densityErrors, float* densitiesPredicted)
        : m_mass(m),
          m_massSquared(m * m),
          m_gravity(gravity),
          m_viscosity(viscosity),
          m_spikyKernel(h),
          m_neighborStarts(neighborStarts),
          m_neighborEnds(neighborEnds),
          m_neighborLists(neighborLists),
          m_positions(positions),
          m_velocities(velocities),
          m_smoothedVelocities(smoothedVelocities),
          m_forces(forces),
          m_densities(densities),
          m_pressures(pressures),
          m_pressureForces(pressureForces),
          m_densityErrors(densityErrors),
          m_densitiesPredicted(densitiesPredicted)
    {
        // Do nothing
    }

    template <typename Index>
    inline CUBBYFLOW_CUDA_DEVICE void operator()(Index i)
    {
        // Initialize buffers
        m_pressures[i] = 0.0f;
        m_pressureForces[i] = make_float4(0, 0, 0, 0);
        m_densityErrors[i] = 0.0f;
        m_densitiesPredicted[i] = m_densities[i];

        // Compute forces
        uint32_t ns = m_neighborStarts[i];
        uint32_t ne = m_neighborEnds[i];
        float4 x_i = m_positions[i];
        float4 v_i = m_velocities[i];
        float d_i = m_densities[i];
        float4 f = m_gravity;
        float w_i = m_mass / d_i * m_spikyKernel(0.0f);
        float weightSum = w_i;
        float4 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj)
        {
            uint32_t j = m_neighborLists[jj];

            float4 r = m_positions[j] - x_i;
            float dist = Length(r);

            if (dist > 0.0f)
            {
                float4 v_j = m_velocities[j];
                float d_j = m_densities[j];

                // Viscosity force
                f += m_viscosity * m_massSquared * (v_j - v_i) / d_j *
                     m_spikyKernel.SecondDerivative(dist);

                // Pseudo viscosity
                float w_j = m_mass / d_j * m_spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        m_forces[i] = f;

        smoothedVelocity /= weightSum;
        m_smoothedVelocities[i] = smoothedVelocity;
    }

 private:
    float m_mass;
    float m_massSquared;
    float4 m_gravity;
    float m_viscosity;
    CUDASPHSpikyKernel3 m_spikyKernel;
    const uint32_t* m_neighborStarts;
    const uint32_t* m_neighborEnds;
    const uint32_t* m_neighborLists;
    const float4* m_positions;
    const float4* m_velocities;
    float4* m_smoothedVelocities;
    float4* m_forces;
    const float* m_densities;
    float* m_pressures;
    float4* m_pressureForces;
    float* m_densitiesPredicted;
    float* m_densityErrors;
};

#define BND_R 0.0f

class TimeIntegration
{
 public:
    TimeIntegration(float dt, float m, float smoothFactor, float3 lower,
                    float3 upper, float4* positions, float4* velocities,
                    float4* newPositions, float4* newVelocities,
                    float4* smoothedVelocities, float4* forces,
                    float4* pressureForces)
        : m_dt(dt),
          m_mass(m),
          m_smoothFactor(smoothFactor),
          m_lower(lower),
          m_upper(upper),
          m_positions(positions),
          m_velocities(velocities),
          m_newPositions(newPositions),
          m_newVelocities(newVelocities),
          m_smoothedVelocities(smoothedVelocities),
          m_forces(forces),
          m_pressureForces(pressureForces)
    {
        // Do nothing
    }

    template <typename Index>
    inline CUBBYFLOW_CUDA_HOST_DEVICE void operator()(Index i)
    {
        float4 x = m_positions[i];
        float4 v = m_velocities[i];
        float4 s = m_smoothedVelocities[i];
        float4 f = m_forces[i];
        float4 pf = m_pressureForces[i];

        v = (1.0f - m_smoothFactor) * v + m_smoothFactor * s;
        v += m_dt * (f + pf) / m_mass;
        x += m_dt * v;

        // TODO: Add proper collider support
        if (x.x > m_upper.x)
        {
            x.x = m_upper.x;
            v.x *= BND_R;
        }
        if (x.x < m_lower.x)
        {
            x.x = m_lower.x;
            v.x *= BND_R;
        }
        if (x.y > m_upper.y)
        {
            x.y = m_upper.y;
            v.y *= BND_R;
        }
        if (x.y < m_lower.y)
        {
            x.y = m_lower.y;
            v.y *= BND_R;
        }

        m_newPositions[i] = x;
        m_newVelocities[i] = v;
    }

 private:
    float m_dt;
    float m_mass;
    float m_smoothFactor;
    float3 m_lower;
    float3 m_upper;
    float4* m_positions;
    float4* m_velocities;
    float4* m_newPositions;
    float4* m_newVelocities;
    float4* m_smoothedVelocities;
    float4* m_forces;
    float4* m_pressureForces;
};

class ComputeDensityError
{
 public:
    inline ComputeDensityError(float m, float h, float targetDensity,
                               float delta, float negativePressureScale,
                               const uint32_t* neighborStarts,
                               const uint32_t* neighborEnds,
                               const uint32_t* neighborLists,
                               const float4* positions, float* pressures,
                               float* densityErrors, float* densitiesPredicted)
        : m_mass(m),
          m_targetDensity(targetDensity),
          m_delta(delta),
          m_negativePressureScale(negativePressureScale),
          m_neighborStarts(neighborStarts),
          m_neighborEnds(neighborEnds),
          m_neighborLists(neighborLists),
          m_positions(positions),
          m_pressures(pressures),
          m_densityErrors(densityErrors),
          m_densitiesPredicted(densitiesPredicted),
          m_stdKernel(h)
    {
        // Do nothing
    }

    template <typename Index>
    inline CUBBYFLOW_CUDA_DEVICE void operator()(Index i)
    {
        uint32_t ns = m_neighborStarts[i];
        uint32_t ne = m_neighborEnds[i];
        float4 x_i = m_positions[i];
        float kernelSum = m_stdKernel(0.f);

        for (uint32_t jj = ns; jj < ne; ++jj)
        {
            uint32_t j = m_neighborLists[jj];

            float4 r = m_positions[j] - x_i;
            float dist = Length(r);

            if (dist > 0.0f)
            {
                kernelSum += m_stdKernel(dist);
            }
        }

        float density = m_mass * kernelSum;
        float densityError = (density - m_targetDensity);
        float pressure = m_delta * densityError;

        if (pressure < 0.0f)
        {
            pressure *= m_negativePressureScale;
            densityError *= m_negativePressureScale;
        }

        m_pressures[i] += pressure;
        m_densitiesPredicted[i] = density;
        m_densityErrors[i] = densityError;
    }

 private:
    float m_mass;
    float m_targetDensity;
    float m_delta;
    float m_negativePressureScale;
    const uint32_t* m_neighborStarts;
    const uint32_t* m_neighborEnds;
    const uint32_t* m_neighborLists;
    const float4* m_positions;
    float* m_pressures;
    float* m_densitiesPredicted;
    float* m_densityErrors;
    CUDASPHStdKernel3 m_stdKernel;
};

class ComputePressureForces
{
 public:
    inline ComputePressureForces(float m, float h,
                                 const uint32_t* neighborStarts,
                                 const uint32_t* neighborEnds,
                                 const uint32_t* neighborLists,
                                 const float4* positions,
                                 float4* pressureForces, const float* densities,
                                 const float* pressures)
        : m_mass(m),
          m_massSquared(m * m),
          m_spikyKernel(h),
          m_neighborStarts(neighborStarts),
          m_neighborEnds(neighborEnds),
          m_neighborLists(neighborLists),
          m_positions(positions),
          m_pressureForces(pressureForces),
          m_densities(densities),
          m_pressures(pressures)
    {
        // Do nothing
    }

    template <typename Index>
    inline CUBBYFLOW_CUDA_HOST_DEVICE void operator()(Index i)
    {
        uint32_t ns = m_neighborStarts[i];
        uint32_t ne = m_neighborEnds[i];

        float4 x_i = m_positions[i];
        float d_i = m_densities[i];
        float p_i = m_pressures[i];

        float4 f = make_float4(0, 0, 0, 0);

        for (uint32_t jj = ns; jj < ne; ++jj)
        {
            uint32_t j = m_neighborLists[jj];

            float4 r = m_positions[j] - x_i;
            float dist = Length(r);

            if (dist > 0.0f)
            {
                float4 dir = r / dist;

                float d_j = m_densities[j];
                float p_j = m_pressures[j];

                // Pressure force
                f -= m_massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     m_spikyKernel.Gradient(dist, dir);
            }
        }

        m_pressureForces[i] = f;
    }

 private:
    float m_mass;
    float m_massSquared;
    CUDASPHSpikyKernel3 m_spikyKernel;
    const uint32_t* m_neighborStarts;
    const uint32_t* m_neighborEnds;
    const uint32_t* m_neighborLists;
    const float4* m_positions;
    float4* m_pressureForces;
    const float* m_densities;
    const float* m_pressures;
};
}  // namespace

void CUDAPCISPHSolver3::OnAdvanceTimeStep(double timeStepInSeconds)
{
    auto sph = SPHSystemData();

    // Build neighbor searcher
    sph->BuildNeighborSearcher();
    sph->BuildNeighborListsAndUpdateDensities();

    auto d = sph->Densities();
    auto p = sph->Pressures();
    const float targetDensity = sph->TargetDensity();

    size_t n = sph->NumberOfParticles();
    float mass = sph->Mass();
    float h = sph->KernelRadius();
    auto ns = sph->NeighborStarts();
    auto ne = sph->NeighborEnds();
    auto nl = sph->NeighborLists();
    auto x = sph->Positions();
    auto v = sph->Velocities();
    auto s = SmoothedVelocities();
    auto f = Forces();

    float dt = static_cast<float>(timeStepInSeconds);
    float factor = dt * PseudoViscosityCoefficient();
    factor = Clamp(factor, 0.0f, 1.0f);

    auto xs = TempPositions();
    auto vs = TempVelocities();
    auto pf = PressureForces();
    auto ds = TempDensities();
    auto de = DensityErrors();

    float delta = ComputeDelta(dt);

    // Initialize buffers and compute non-pressure forces
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),
        InitializeBuffersAndComputeForces(
            mass, h, ToFloat4(Gravity(), 0.0f), ViscosityCoefficient(),
            ns.data(), ne.data(), nl.data(), x.data(), v.data(), s.data(),
            f.data(), d.data(), p.data(), pf.data(), de.data(), ds.data()));

    // Prediction-correction
    auto lower = ToFloat3(Container().lowerCorner);
    auto upper = ToFloat3(Container().upperCorner);

    for (unsigned int k = 0; k < m_maxNumberOfIterations; ++k)
    {
        // Predict velocity / position and resolve collisions
        thrust::for_each(thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(n),
                         TimeIntegration(dt, mass, 0.0f, lower, upper, x.data(),
                                         v.data(), xs.data(), vs.data(),
                                         s.data(), f.data(), pf.data()));

        // Compute pressure from density error
        thrust::for_each(thrust::counting_iterator<size_t>(0),
                         thrust::counting_iterator<size_t>(n),
                         ComputeDensityError(mass, h, targetDensity, delta,
                                             NegativePressureScale(), ns.data(),
                                             ne.data(), nl.data(), xs.data(),
                                             p.data(), de.data(), ds.data()));

        // Compute pressure gradient force
        thrust::for_each(
            thrust::counting_iterator<size_t>(0),
            thrust::counting_iterator<size_t>(n),
            ComputePressureForces(mass, h, ns.data(), ne.data(), nl.data(),
                                  x.data(), pf.data(), ds.data(), p.data()));
    }

    // Accumulate pressure force and time-integrate
    thrust::for_each(
        thrust::counting_iterator<size_t>(0),
        thrust::counting_iterator<size_t>(n),
        TimeIntegration(dt, mass, factor, lower, upper, x.data(), v.data(),
                        x.data(), v.data(), s.data(), f.data(), pf.data()));
}
