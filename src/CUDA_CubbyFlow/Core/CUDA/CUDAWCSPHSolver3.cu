#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Core/CUDA/CUDASPHKernels3.hpp>
#include <Core/CUDA/CUDAWCSPHSolver3.hpp>

using namespace CubbyFlow;

namespace
{
inline __device__ float ComputePressureFromEos(float density,
                                               float targetDensity,
                                               float eosScale,
                                               float eosExponent,
                                               float negativePressureScale)
{
    // Equation of state
    // (http://www.ifi.uzh.ch/vmml/publications/pcisph/pcisph.pdf)
    float p = eosScale / eosExponent *
              (powf((density / targetDensity), eosExponent) - 1.0f);

    // Negative pressure scaling
    if (p < 0)
    {
        p *= negativePressureScale;
    }

    return p;
}

__global__ void ComputePressureKernel(float targetDensity, float eosScale,
                                      float eosExponent,
                                      float negativePressureScale,
                                      const float* densities, size_t n,
                                      float* pressures)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        pressures[i] =
            ComputePressureFromEos(densities[i], targetDensity, eosScale,
                                   eosExponent, negativePressureScale);
    }
}

__global__ void ComputeForcesKernel(
    float mass, float4 gravity, float viscosity,
    CUDASPHSpikyKernel3 spikyKernel, const uint32_t* neighborStarts,
    const uint32_t* neighborEnds, const uint32_t* neighborLists,
    const float4* positions, const float4* velocities, const float* densities,
    const float* pressures, size_t n, float4* smoothedVelocities,
    float4* forces)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        uint32_t ns = neighborStarts[i];
        uint32_t ne = neighborEnds[i];

        float4 x_i = positions[i];
        float4 v_i = velocities[i];
        float d_i = densities[i];
        float p_i = pressures[i];
        float4 f = gravity;

        float massSquared = mass * mass;

        float w_i = mass / d_i * spikyKernel(0.0f);
        float weightSum = w_i;
        float4 smoothedVelocity = w_i * v_i;

        for (uint32_t jj = ns; jj < ne; ++jj)
        {
            uint32_t j = neighborLists[jj];

            float4 r = positions[j] - x_i;
            float dist = Length(r);

            if (dist > 0.0f)
            {
                float4 dir = r / dist;

                float4 v_j = velocities[j];
                float d_j = densities[j];
                float p_j = pressures[j];

                // Pressure force
                f -= massSquared * (p_i / (d_i * d_i) + p_j / (d_j * d_j)) *
                     spikyKernel.Gradient(dist, dir);

                // Viscosity force
                f += viscosity * massSquared * (v_j - v_i) / d_j *
                     spikyKernel.SecondDerivative(dist);

                // Pseudo viscosity
                float w_j = mass / d_j * spikyKernel(dist);
                weightSum += w_j;
                smoothedVelocity += w_j * v_j;
            }
        }

        forces[i] = f;

        smoothedVelocity /= weightSum;
        smoothedVelocities[i] = smoothedVelocity;
    }
}

#define BND_R 0.0f

__global__ void TimeIntegrationKernel(float dt, float mass, float smoothFactor,
                                      float3 lower, float3 upper,
                                      const float4* smoothedVelocities,
                                      const float4* forces, size_t n,
                                      float4* positions, float4* velocities)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        float4 x = positions[i];
        float4 v = velocities[i];
        float4 s = smoothedVelocities[i];
        float4 f = forces[i];

        v = (1.0f - smoothFactor) * v + smoothFactor * s;
        v += dt * f / mass;
        x += dt * v;

        // TODO: Add proper collider support
        if (x.x > upper.x)
        {
            x.x = upper.x;
            v.x *= BND_R;
        }
        if (x.x < lower.x)
        {
            x.x = lower.x;
            v.x *= BND_R;
        }
        if (x.y > upper.y)
        {
            x.y = upper.y;
            v.y *= BND_R;
        }
        if (x.y < lower.y)
        {
            x.y = lower.y;
            v.y *= BND_R;
        }
        if (x.z > upper.z)
        {
            x.z = upper.z;
            v.z *= BND_R;
        }
        if (x.z < lower.z)
        {
            x.z = lower.z;
            v.z *= BND_R;
        }

        positions[i] = x;
        velocities[i] = v;
    }
}
}  // namespace

void CUDAWCSPHSolver3::OnAdvanceTimeStep(double timeStepInSeconds)
{
    auto sph = SPHSystemData();
    size_t n = sph->NumberOfParticles();

    // Build neighbor searcher
    sph->BuildNeighborSearcher();
    sph->BuildNeighborListsAndUpdateDensities();

    // Compute pressure
    auto d = sph->Densities();
    auto p = sph->Pressures();
    const float targetDensity = sph->TargetDensity();
    const float eosScale =
        targetDensity * Square(SpeedOfSound()) / m_eosExponent;

    unsigned int numBlocks, numThreads;
    CUDAComputeGridSize(static_cast<unsigned int>(n), 256, numBlocks,
                        numThreads);

    ComputePressureKernel<<<numBlocks, numThreads>>>(
        targetDensity, eosScale, m_eosExponent, NegativePressureScale(),
        d.data(), n, p.data());

    // Compute pressure/viscosity forces and smoothed velocity
    float mass = sph->Mass();
    float h = sph->KernelRadius();
    auto ns = sph->NeighborStarts();
    auto ne = sph->NeighborEnds();
    auto nl = sph->NeighborLists();
    auto x = sph->Positions();
    auto v = sph->Velocities();
    auto s = SmoothedVelocities();
    auto f = Forces();

    ComputeForcesKernel<<<numBlocks, numThreads>>>(
        mass, ToFloat4(Gravity(), 0.0f), ViscosityCoefficient(),
        CUDASPHSpikyKernel3(h), ns.data(), ne.data(), nl.data(), x.data(),
        v.data(), d.data(), p.data(), n, s.data(), f.data());

    // Time-integration
    float dt = static_cast<float>(timeStepInSeconds);
    float factor = dt * PseudoViscosityCoefficient();
    factor = Clamp(factor, 0.0f, 1.0f);
    auto lower = ToFloat3(Container().lowerCorner);
    auto upper = ToFloat3(Container().upperCorner);

    TimeIntegrationKernel<<<numBlocks, numThreads>>>(dt, mass, factor, lower,
                                                     upper, s.data(), f.data(),
                                                     n, x.data(), v.data());
}