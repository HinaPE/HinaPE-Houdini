#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Core/CUDA/CUDAAlgorithms.hpp>
#include <Core/CUDA/CUDAPointHashGridSearcher2.hpp>

#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/sort.h>

using namespace CubbyFlow;

namespace
{
__global__ void InitializeIndexTables(uint32_t* startIndexTable,
                                      uint32_t* endIndexTable, size_t n)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        startIndexTable[i] = 0xffffffff;
        endIndexTable[i] = 0xffffffff;
    }
}

__global__ void InitializePointAndKeys(
    CUDAPointHashGridSearcher2::HashUtils hashUtils, const float2* points,
    size_t n, uint32_t* sortedIndices, uint32_t* keys)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        sortedIndices[i] = i;
        keys[i] = hashUtils.GetHashKeyFromPosition(points[i]);
    }
}

__global__ void BuildTables(const uint32_t* keys, size_t n,
                            uint32_t* startIndexTable, uint32_t* endIndexTable)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && i > 0)
    {
        uint32_t k = keys[i];
        uint32_t kLeft = keys[i - 1];
        if (k > kLeft)
        {
            startIndexTable[k] = i;
            endIndexTable[kLeft] = i;
        }
    }
}
}  // namespace

CUDAPointHashGridSearcher2::CUDAPointHashGridSearcher2(uint32_t resolutionX,
                                                       uint32_t resolutionY,
                                                       float gridSpacing)
    : m_gridSpacing{ gridSpacing }
{
    m_resolution.x = std::max(resolutionX, 1u);
    m_resolution.y = std::max(resolutionY, 1u);

    m_startIndexTable.Resize(m_resolution.x * m_resolution.y, 0xffffffff);
    m_endIndexTable.Resize(m_resolution.x * m_resolution.y, 0xffffffff);
}

void CUDAPointHashGridSearcher2::Build(
    const ConstCUDAArrayView1<float2>& points)
{
    // Allocate/reset memory chuncks
    size_t numberOfPoints = points.Length();
    if (numberOfPoints == 0)
    {
        return;
    }

    m_points = points;

    // Initialize index tables
    size_t numberOfGrids = m_startIndexTable.Length();
    unsigned int numBlocks, numThreads;
    CUDAComputeGridSize(static_cast<unsigned int>(numberOfPoints), 256,
                        numBlocks, numThreads);

    InitializeIndexTables<<<numBlocks, numThreads>>>(
        m_startIndexTable.data(), m_endIndexTable.data(),
        m_startIndexTable.Length());

    // Initialize indices array and generate hash key for each point
    m_keys.Resize(numberOfPoints);
    m_sortedIndices.Resize(numberOfPoints);

    CUDAComputeGridSize(static_cast<unsigned int>(numberOfPoints), 256,
                        numBlocks, numThreads);

    CUDAPointHashGridSearcher2::HashUtils hashUtils(m_gridSpacing,
                                                    m_resolution);

    InitializePointAndKeys<<<numBlocks, numThreads>>>(
        hashUtils, m_points.data(), m_points.Length(), m_sortedIndices.data(),
        m_keys.data());

    // Sort indices/points/key based on hash key
    thrust::device_ptr<uint32_t> keysBegin(m_keys.data());
    thrust::device_ptr<uint32_t> keysEnd = keysBegin + m_keys.Length();
    thrust::device_ptr<float2> pointsBegin(m_points.data());
    thrust::device_ptr<uint32_t> sortedIndicesBegin(m_sortedIndices.data());
    thrust::sort_by_key(keysBegin, keysEnd,
                        thrust::make_zip_iterator(thrust::make_tuple(
                            pointsBegin, sortedIndicesBegin)));

    // Now _points and _keys are sorted by points' hash key values.
    // Let's fill in start/end index table with _keys.
    // Assume that _keys array looks like:
    // [5|8|8|10|10|10]
    // Then _startIndexTable and _endIndexTable should be like:
    // [.....|0|...|1|..|3|..]
    // [.....|1|...|3|..|6|..]
    //       ^5    ^8   ^10
    // So that _endIndexTable[i] - _startIndexTable[i] is the number points in
    // i-th table bucket.

    m_startIndexTable[m_keys[0]] = 0;
    m_endIndexTable[m_keys[numberOfPoints - 1]] =
        static_cast<uint32_t>(numberOfPoints);

    CUDAComputeGridSize(static_cast<unsigned int>(numberOfPoints), 256,
                        numBlocks, numThreads);

    BuildTables<<<numBlocks, numThreads>>>(m_keys.data(), numberOfPoints,
                                           m_startIndexTable.data(),
                                           m_endIndexTable.data());
}

void CUDAPointHashGridSearcher2::Set(const CUDAPointHashGridSearcher2& other)
{
    m_gridSpacing = other.m_gridSpacing;
    m_resolution = other.m_resolution;
    m_points = other.m_points;
    m_keys = other.m_keys;
    m_startIndexTable = other.m_startIndexTable;
    m_endIndexTable = other.m_endIndexTable;
    m_sortedIndices = other.m_sortedIndices;
}