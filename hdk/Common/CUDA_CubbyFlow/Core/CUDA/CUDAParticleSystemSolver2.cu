#include "hip/hip_runtime.h"
// This code is based on Jet framework.
// Copyright (c) 2018 Doyub Kim
// CubbyFlow is voxel-based fluid simulation engine for computer games.
// Copyright (c) 2020 CubbyFlow Team
// Core Part: Chris Ohk, Junwoo Hwang, Jihong Sin, Seungwoo Yoo
// AI Part: Dongheon Cho, Minseo Kim
// We are making my contributions/submissions to this project solely in our
// personal capacity and are not conveying any rights to any intellectual
// property of any third parties.

#include <Core/CUDA/CUDAParticleSystemSolver2.hpp>
#include <Core/CUDA/CUDAUtils.hpp>

using namespace CubbyFlow;

namespace
{
__global__ void AdvanceTimeStepKernel(float dt, float2 gravity,
                                      const float2* positions,
                                      const float2* velocities, size_t n,
                                      float2* newPositions,
                                      float2* newVelocities)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        // Time integration
        float2 v1 = velocities[i] + dt * gravity;
        float2 p1 = positions[i] + dt * v1;

        // Collision handling
        if (p1.y < 0.0f)
        {
            p1.y = 0.0f;

            if (v1.y < 0.0f)
            {
                v1.y *= -1.0f;
            }
        }

        newPositions[i] = p1;
        newVelocities[i] = v1;
    }
}

}  // namespace

void CUDAParticleSystemSolver2::OnAdvanceTimeStep(double timeStepInSeconds)
{
    UpdateCollider(timeStepInSeconds);
    UpdateEmitter(timeStepInSeconds);

    auto particles = ParticleSystemData();
    const size_t n = particles->NumberOfParticles();
    auto curPos = particles->Positions();
    auto curVel = particles->Velocities();
    const auto dt = static_cast<float>(timeStepInSeconds);
    const auto g = ToFloat2(Gravity());

    unsigned int numBlocks, numThreads;
    CUDAComputeGridSize(static_cast<unsigned>(n), 256, numBlocks, numThreads);
    AdvanceTimeStepKernel<<<numBlocks, numThreads>>>(
        dt, g, curPos.data(), curVel.data(), n, curPos.data(), curVel.data());
}