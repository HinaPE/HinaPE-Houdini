#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

__global__ void cuda_hello()
{
	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int tnum = gridDim.x * blockDim.x;
	printf("Hello World from GPU! %d, %d, %d | %d, %d, %d | %d, %d, %d | %d, %d, %d\n", threadIdx.x, threadIdx.y, threadIdx.z, gridDim.x, gridDim.y, gridDim.z, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z);
}

int main()
{
	cuda_hello<<<3, 3>>>(); // <<< grid_dim， block_dim >>> // block num, thread num
	hipDeviceSynchronize();
	return 0;
}
